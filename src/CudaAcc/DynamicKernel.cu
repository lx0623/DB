#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "DynamicKernel.h"
#include "AriesColumnDataIterator.hxx"
#include "utils/string_util.h"
#include "server/Configuration.h"
#include "CpuTimer.h"

BEGIN_ARIES_ACC_NAMESPACE

void AriesDynamicKernelManager::SetModuleAccessed( const string& code )
{
    //update lru
    auto it = std::find( m_moduleCodesLru.begin(), m_moduleCodesLru.end(), code );
    assert( it != m_moduleCodesLru.end() );
    string val = *it;
    m_moduleCodesLru.erase( it );
    m_moduleCodesLru.push_front( val );
}

void AriesDynamicKernelManager::AddModule( const string& code, AriesCUModuleInfoSPtr moduleInfo )
{
    unique_lock< mutex > lock( m_mutex );
    if( m_modules.find( code ) == m_modules.end() )
    {
        m_modules.insert( { code, moduleInfo } );
        m_moduleCodesLru.push_front( code );
        RemoveOldModulesIfNecessary();
    }
    else
        SetModuleAccessed( code );
}

void AriesDynamicKernelManager::RemoveOldModulesIfNecessary()
{
    assert( m_modules.size() == m_moduleCodesLru.size() );
    int needRemoveCount = m_moduleCodesLru.size() - LRU_COUNT;
    if( needRemoveCount > 0 )
    {
        while( needRemoveCount-- )
        {
            m_modules.erase( m_moduleCodesLru.back() );
            m_moduleCodesLru.pop_back();
        }
    }
}

AriesCUModuleInfoSPtr AriesDynamicKernelManager::FindModule( const string& code )
{
    unique_lock< mutex > lock( m_mutex );
    AriesCUModuleInfoSPtr result;
    auto it = m_modules.find( code );
    if( it != m_modules.end() )
    {
        result = it->second;
        SetModuleAccessed( code );
    }
    return result;
}

AriesCUModuleInfoSPtr AriesDynamicKernelManager::CompileKernels( const AriesDynamicCodeInfo& code )
{
    AriesCUModuleInfoSPtr moduleInfo = FindModule( code.KernelCode );
    if( moduleInfo )
        return moduleInfo;
    else
        moduleInfo = std::make_shared< AriesCUModuleInfo >();
    vector< CUmoduleSPtr > result;
    // m_ctx->timer_begin();
    hiprtcProgram prog;
    NVRTC_SAFE_CALL( hiprtcCreateProgram(&prog,         // prog
            code.KernelCode.c_str(),// buffer
            0,// name
            0,// numHeaders
            NULL,// headers
            NULL) );// includeNames
    static std::string include_path( "-I " + aries_utils::get_current_work_directory() + "/include" );
    // Compile the program for compute_30 with fmad disabled.
    // we assmue all device has same compute capability.
    aries::Configuartion& config = aries::Configuartion::GetInstance();
    string computeVersionParam = "--gpu-architecture=compute_" + std::to_string( config.GetComputeVersionMajor() ) + std::to_string( config.GetComputeVersionMinor() );
    const char *opts[] =
    { computeVersionParam.c_str(), "--relocatable-device-code=true", "--std=c++14", "--define-macro=__WORDSIZE=64", include_path.c_str() };
    hiprtcResult compileResult = hiprtcCompileProgram( prog,  // prog
            5,     // numOptions
            opts ); // options
    // LOG( INFO ) << "hiprtcCompileProgram gpu time: " << m_ctx->timer_end();

    // Obtain compilation log from the program.
    size_t logSize;
    NVRTC_SAFE_CALL( hiprtcGetProgramLogSize( prog, &logSize ) );
    char *log = new char[logSize];
    NVRTC_SAFE_CALL( hiprtcGetProgramLog( prog, log ) );
    LOG( INFO ) << log;
    delete[] log;
    if( compileResult != HIPRTC_SUCCESS )
    {
        LOG( INFO ) << "dyn compile error\n";
        return moduleInfo;
    }
    // Obtain PTX from the program.
    size_t ptxSize;
    NVRTC_SAFE_CALL( hiprtcGetCodeSize( prog, &ptxSize ) );
    LOG( INFO ) << "ptx size is:" << ptxSize << " bytes" << std::endl;
    char *ptx = new char[ptxSize];
    NVRTC_SAFE_CALL( hiprtcGetCode( prog, ptx ) );
    //std::cout<<ptx<<std::endl;
    // Destroy the program.
    NVRTC_SAFE_CALL( hiprtcDestroyProgram( &prog ) );
    // Load the generated PTX and get a handle to the kernel.

    int32_t oldDeviceId;// = m_ctx->active_device_id();
    hipGetDevice(&oldDeviceId);

    hipDevice_t cuDevice;
    hipCtx_t context;
    CU_SAFE_CALL( hipDeviceGet( &cuDevice, oldDeviceId ) );
    CU_SAFE_CALL( hipDevicePrimaryCtxRetain( &context, cuDevice ) );
    CU_SAFE_CALL( hipCtxPushCurrent( context ) );

    hiprtcLinkState linkState;
    CU_SAFE_CALL( hiprtcLinkCreate( 0, 0, 0, &linkState ) );

    // m_ctx->timer_begin();
    static std::string library_path( aries_utils::get_current_work_directory() + "/lib/libariesdatatype.a" );
    CU_SAFE_CALL( hiprtcLinkAddFile( linkState, hipJitInputLibrary, library_path.c_str(), 0, 0, 0 ) );
    // LOG( INFO ) << "hiprtcLinkAddFile gpu time: " << m_ctx->timer_end();

    // m_ctx->timer_begin();
    CU_SAFE_CALL( hiprtcLinkAddData( linkState, hipJitInputPtx, ( void * )ptx, ptxSize, 0, 0, 0, 0 ) );
    // LOG( INFO ) << "hiprtcLinkAddData gpu time: " << m_ctx->timer_end();

    delete[] ptx;
    size_t cubinSize;
    void *cubin;
    CU_SAFE_CALL( hiprtcLinkComplete( linkState, &cubin, &cubinSize ) );

    int deviceCount;
    hipGetDeviceCount( &deviceCount );
    for( int deviceId = 0; deviceId < deviceCount; ++deviceId )
    {
        CUmoduleSPtr module( new hipModule_t, []( hipModule_t* p )
        {   CU_SAFE_CALL( hipModuleUnload( *p ) ); delete p;} );
        hipSetDevice( deviceId );
        CU_SAFE_CALL( hipModuleLoadData( module.get(), cubin ) );
        result.push_back( module );
    }

    hipSetDevice( oldDeviceId );

    CU_SAFE_CALL( hiprtcLinkDestroy( linkState ) );
    CU_SAFE_CALL( hipCtxPopCurrent( nullptr ) );
    CU_SAFE_CALL( hipDevicePrimaryCtxRelease( cuDevice ) );

    moduleInfo->Modules = std::move( result );
    moduleInfo->FunctionKeyNameMapping = code.FunctionKeyNameMapping;
    AddModule( code.KernelCode, moduleInfo );
    return moduleInfo;
}

void AriesDynamicKernelManager::CallKernel( const vector< CUmoduleSPtr >& modules,
                                            const char* functionName,
                                            const AriesColumnDataIterator *input,
                                            const index_t *leftIndices,
                                            const index_t *rightIndices,
                                            int tupleNum,
                                            const int8_t** constValues,
                                            const vector< AriesDynamicCodeComparator >& items,
                                            int8_t *output ) const
{
    ARIES_ASSERT( !modules.empty(), "CUDA module for " + string( functionName ) + " is empty, no dynamic code or no compiling for it?" );
    // m_ctx->timer_begin();
    hipDevice_t dev;
    hipGetDevice( &dev );//m_ctx->active_device_id();
    ARIES_ASSERT( dev < modules.size(),
            "CUDA module for device " + std::to_string( dev ) + " is empty, no dynamic code or no compiling for it?" );
    CUmoduleSPtr module = modules[dev];

    AriesArraySPtr< CallableComparator* > comparators = CreateInComparators( module, items );
    CallableComparator** operators = comparators->GetData();
    hipFunction_t kernel;
    CU_SAFE_CALL( hipModuleGetFunction( &kernel, *module, functionName ) );

    void *args[] =
    { &input, &leftIndices, &rightIndices, &tupleNum, &constValues, &operators, &output };

    int numThreads = 256;
    int numBlocks = ( tupleNum + numThreads - 1 ) / numThreads;
    hipError_t res = hipModuleLaunchKernel( kernel, numBlocks, 1, 1,    // grid dim
            numThreads, 1, 1,   // block dim
            0, NULL,             // shared mem and stream
            args, 0 );           // arguments
    CU_SAFE_CALL( res );
    CU_SAFE_CALL( hipCtxSynchronize() );
    DestroyComparators( comparators );
    // LOG( INFO ) << "CallKernel gpu time: " << m_ctx->timer_end();
}

void AriesDynamicKernelManager::CallKernel( const vector< CUmoduleSPtr >& modules,
                                            const char* functionName,
                                            const AriesColumnDataIterator *input,
                                            size_t leftCount,
                                            size_t rightCount,
                                            size_t tupleNum,
                                            int* left_unmatched_flag,
                                            int* right_unmatched_flag,
                                            const int8_t** constValues,
                                            const vector< AriesDynamicCodeComparator >& items,
                                            int *left_output,
                                            int *right_output,
                                            unsigned long long int* output_count ) const
{
    ARIES_ASSERT( !modules.empty(), "CUDA module for " + string( functionName ) + " is empty, no dynamic code or no compiling for it?" );
    // m_ctx->timer_begin();
    hipDevice_t dev; // = m_ctx->active_device_id();
    hipGetDevice( &dev );
    ARIES_ASSERT( dev < modules.size(),
            "CUDA module for device " + std::to_string( dev ) + " is empty, no dynamic code or no compiling for it?" );
    CUmoduleSPtr module = modules[dev];

    AriesArraySPtr< CallableComparator* > comparators = CreateInComparators( module, items );
    CallableComparator** operators = comparators->GetData();
    hipFunction_t kernel;
    CU_SAFE_CALL( hipModuleGetFunction( &kernel, *module, functionName ) );

    void *args[] =
    { &input, &leftCount, &rightCount, &tupleNum, &left_unmatched_flag, &right_unmatched_flag, &constValues, &operators, &left_output, &right_output,
            &output_count };

    int numThreads = 256;
    int numBlocks = ( tupleNum + numThreads - 1 ) / numThreads;
    hipError_t res = hipModuleLaunchKernel( kernel, numBlocks, 1, 1,    // grid dim
            numThreads, 1, 1,   // block dim
            0, NULL,             // shared mem and stream
            args, 0 );           // arguments
    CU_SAFE_CALL( res );
    CU_SAFE_CALL( hipCtxSynchronize() );
    DestroyComparators( comparators );
    // LOG( INFO ) << "CallKernel gpu time: " << m_ctx->timer_end();
}

void AriesDynamicKernelManager::CallKernel( const vector< CUmoduleSPtr >& modules,
                                            const char* functionName,
                                            const AriesColumnDataIterator *input,
                                            const int32_t* associated,
                                            const int32_t* groups,
                                            const int32_t *group_size_prefix_sum,
                                            int32_t group_count,
                                            int32_t tupleNum,
                                            const int8_t** constValues,
                                            const vector< AriesDynamicCodeComparator >& items,
                                            int8_t *output ) const
{
    ARIES_ASSERT( !modules.empty(), "CUDA module for " + string( functionName ) + " is empty, no dynamic code or no compiling for it?" );
    // m_ctx->timer_begin();
    hipDevice_t dev; // = m_ctx->active_device_id();
    hipGetDevice( &dev );
    ARIES_ASSERT( dev < modules.size(),
            "CUDA module for device " + std::to_string( dev ) + " is empty, no dynamic code or no compiling for it?" );
    CUmoduleSPtr module = modules[dev];

    AriesArraySPtr< CallableComparator* > comparators = CreateInComparators( module, items );
    CallableComparator** operators = comparators->GetData();
    hipFunction_t kernel;
    CU_SAFE_CALL( hipModuleGetFunction( &kernel, *module, functionName ) );

    void *args[] =
    { &input, &associated, &groups, &group_size_prefix_sum, &group_count, &tupleNum, &constValues, &operators, &output };

    int numThreads = 256;
    int numBlocks = ( tupleNum + numThreads - 1 ) / numThreads;
    hipError_t res = hipModuleLaunchKernel( kernel, numBlocks, 1, 1,    // grid dim
            numThreads, 1, 1,   // block dim
            0, NULL,             // shared mem and stream
            args, 0 );           // arguments
    CU_SAFE_CALL( res );
    CU_SAFE_CALL( hipCtxSynchronize() );
    DestroyComparators( comparators );
    // LOG( INFO ) << "CallKernel gpu time: " << m_ctx->timer_end();
}

void AriesDynamicKernelManager::CallKernel( const vector< CUmoduleSPtr >& modules,
                                            const char *functionName,
                                            const AriesColumnDataIterator *input,
                                            int tupleNum,
                                            const int8_t** constValues,
                                            const vector< AriesDynamicCodeComparator >& items,
                                            int8_t *output ) const
{
    ARIES_ASSERT( !modules.empty(), "CUDA module for " + string( functionName ) + " is empty, no dynamic code or no compiling for it?" );
//     m_ctx->timer_begin();
    #ifdef ARIES_PROFILE
            aries::CPU_Timer t;
            t.begin();
    #endif
    hipDevice_t dev; // = m_ctx->active_device_id();
    hipGetDevice(&dev);
    ARIES_ASSERT( dev < modules.size(),
            "CUDA module for device " + std::to_string( dev ) + " is empty, no dynamic code or no compiling for it?" );
    CUmoduleSPtr module = modules[dev];
    AriesArraySPtr< CallableComparator* > comparators = CreateInComparators( module, items );
    CallableComparator** operators = comparators->GetData();
    hipFunction_t kernel;
    CU_SAFE_CALL( hipModuleGetFunction( &kernel, *module, functionName ) );

    void *args[] =
    { &input, &tupleNum, &constValues, &operators, &output };

    int numThreads = 256;
    int numBlocks = ( tupleNum + numThreads - 1 ) / numThreads;
    hipError_t res = hipModuleLaunchKernel( kernel, numBlocks, 1, 1,    // grid dim
            numThreads, 1, 1,   // block dim
            0, NULL,             // shared mem and stream
            args, 0 );           // arguments
    CU_SAFE_CALL( res );
    CU_SAFE_CALL( hipCtxSynchronize() );
    DestroyComparators( comparators );
#ifdef ARIES_PROFILE
    long long kernel_time = t.end();
    LOG( INFO ) << "CallKernel gpu time: " << kernel_time;
#endif
//     LOG( INFO ) << "CallKernel gpu time: " << m_ctx->timer_end();
}


__global__ void DestroyComparatorsKernel(CallableComparator** data, size_t itemCount)
{
        int tid = threadIdx.x + blockDim.x * blockIdx.x;
        if (tid<itemCount) delete data[tid];
}

void AriesDynamicKernelManager::DestroyComparators( AriesArraySPtr< CallableComparator* > comparators ) const
{
    CallableComparator** data = comparators->GetData();
    size_t itemCount = comparators->GetItemCount();
    if( itemCount > 0 )
    {
        int block = (itemCount + 31) / 32;
        DestroyComparatorsKernel<<<block, 32>>>(data, itemCount);
        // auto k = [=] ARIES_DEVICE(int index)
        // {
        //     for( int i = 0; i < itemCount; ++i )
        //     delete data[i];
        // };
        // transform< launch_box_t< arch_52_cta< 1, 1 > > >( k, 1, *m_ctx );
        // m_ctx->synchronize();
    }
}

AriesArraySPtr< CallableComparator* > AriesDynamicKernelManager::CreateInComparators( CUmoduleSPtr module,
        const vector< AriesDynamicCodeComparator >& items ) const
{
    size_t itemCount = items.size();
    AriesArraySPtr< CallableComparator* > result = std::make_shared< AriesArray< CallableComparator* > >( itemCount );

    if( itemCount > 0 )
    {
        AriesManagedArray< AriesKernelParamInfo > params( itemCount );
        AriesKernelParamInfo* paramData = params.GetData();
        int i = 0;
        for( const auto& item : items )
        {
            auto& info = paramData[i++];
            info.Data = item.LiteralBuffer->GetData();
            info.Count = item.LiteralBuffer->GetItemCount();
            info.Len = item.Type.GetDataTypeSize();
            info.Type = item.Type.DataType.ValueType;
            info.HasNull = item.Type.HasNull;
            info.OpType = item.OpType;
        }
        CallableComparator** comparators = result->GetData();

        hipFunction_t kernel;
        CU_SAFE_CALL( hipModuleGetFunction( &kernel, *module, "KernelCreateInComparator" ) );

        params.PrefetchToGpu();
        void *args[] =
        { &paramData, &itemCount, &comparators };
        hipError_t res = hipModuleLaunchKernel( kernel, 1, 1, 1,    // grid dim
                1, 1, 1,   // block dim
                0, NULL,             // shared mem and stream
                args, 0 );           // arguments
        CU_SAFE_CALL( res );
        CU_SAFE_CALL( hipCtxSynchronize() );
    }

    return result;
}

END_ARIES_ACC_NAMESPACE
