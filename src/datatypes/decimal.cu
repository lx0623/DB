#include "hip/hip_runtime.h"
/*
 * decimal.cxx
 *
 *  Created on: 2019年6月26日
 *      Author: david
 */
#include "decimal.hxx"
#include <cassert>
#include <cmath>
#include "AriesDataTypeUtil.hxx"


BEGIN_ARIES_ACC_NAMESPACE

#define FIX_INTG_FRAC_ERROR(len, intg1, frac1, error)       \
    do                                                      \
    {                                                       \
        if (intg1+frac1 > (len))                            \
        {                                                   \
            if (intg1 > (len))                              \
            {                                               \
                intg1=(len);                                \
                frac1=0;                                    \
                error=ERR_OVER_FLOW;                        \
            }                                               \
            else                                            \
            {                                               \
                frac1=(len)-intg1;                          \
                error=ERR_TRUNCATED;                        \
            }                                               \
        }                                                   \
        else                                                \
        {                                                   \
            error=ERR_OK;                                   \
        }                                                   \
    } while(0)

#define FIX_TAGET_INTG_FRAC_ERROR(len, intg1, frac1, error) \
    do                                                      \
    {                                                       \
        if (intg1+frac1 > (len))                            \
        {                                                   \
            if (frac1 > (len))                              \
            {                                               \
                intg1=(len);                                \
                frac1=0;                                    \
                error=ERR_OVER_FLOW;                        \
            }                                               \
            else                                            \
            {                                               \
                intg1=(len)-frac1;                          \
                error=ERR_TRUNCATED;                        \
            }                                               \
        }                                                   \
        else                                                \
        {                                                   \
            error=ERR_OK;                                   \
        }                                                   \
    } while(0)

#define SET_PREC_SCALE_VALUE(t, d0, d1, d2) (t = (d1 != d2 ? d1 * DIG_PER_INT32 : d0))

    ARIES_HOST_DEVICE_NO_INLINE Decimal::Decimal() : Decimal(DEFAULT_PRECISION, DEFAULT_SCALE) {}

//    ARIES_HOST_DEVICE_NO_INLINE Decimal::Decimal( const Decimal& d )
//    {
//        intg = d.intg;
//        frac = d.frac;
//        mode = d.mode;
//        error = d.error;
//        for( int i = 0; i < NUM_TOTAL_DIG; i++ )
//        {
//            values[i] = d.values[i];
//        }
//    }

    ARIES_HOST_DEVICE_NO_INLINE Decimal::Decimal(uint32_t precision, uint32_t scale) : Decimal(precision, scale, (uint32_t) ARIES_MODE_EMPTY) {
    }

    ARIES_HOST_DEVICE_NO_INLINE Decimal::Decimal(uint32_t precision, uint32_t scale, uint32_t m) {
        initialize(precision - scale, scale, m);
    }

    ARIES_HOST_DEVICE_NO_INLINE Decimal::Decimal(uint32_t precision, uint32_t scale, const char s[]) : Decimal( precision, scale, ARIES_MODE_EMPTY, s) {
    }

    ARIES_HOST_DEVICE_NO_INLINE Decimal::Decimal(uint32_t precision, uint32_t scale, uint32_t m, const char s[] ) {
        initialize(precision - scale, scale, m);
        Decimal d(s);
        cast(d);
    }

    ARIES_HOST_DEVICE_NO_INLINE Decimal::Decimal(const CompactDecimal *compact, uint32_t precision, uint32_t scale, uint32_t m) {
        initialize(precision - scale, scale, m);
        SignPos signPos;
        int fracBits = GetDecimalNeedBits(frac);
        int intgBits = GetDecimalNeedBits(intg);
        int realFracBytes = NEEDBYTES(fracBits);
        int realIntgBytes = NEEDBYTES(intgBits);
        if (HAS_FREE_BIT(intgBits)) {
            signPos = INTG_PART;
        } else if (HAS_FREE_BIT(fracBits)) {
            signPos = FRAC_PART;
        } else {
            signPos = ADDITIONAL_PART;
        }
        int sign = 0;
        //handle frag part
        int fracInts = NEEDELEMENTS(frac);
        if (realFracBytes) {
            aries_memcpy((char *)(values + (NUM_TOTAL_DIG - fracInts) ), compact->data + realIntgBytes, realFracBytes);
            if (signPos == FRAC_PART) {
                char *temp = ((char *)(values + INDEX_LAST_DIG));
                if (GET_COMPACT_BYTES(realFracBytes) == realFracBytes) {
                    // <= 3 bytes only
                    temp += GET_COMPACT_BYTES(realFracBytes) - 1;
                } else {
                    // >=4 bytes, have one sort
                    if(GET_COMPACT_BYTES(realFracBytes) != 0)
                        temp -= 1;
                    else
                        temp += 3;
                }
                sign = GET_SIGN_FROM_BIT(*temp);
                *temp = *temp & 0x7f;
            }
            if (GET_COMPACT_BYTES(realFracBytes)) {
                values[INDEX_LAST_DIG] = values[INDEX_LAST_DIG] * GetPowers10( DIG_PER_INT32 - frac % DIG_PER_INT32);
            }
        }
        //handle intg part
        if (realIntgBytes) {
            int wholeInts = GET_WHOLE_INTS(realIntgBytes);
            int compactPart = GET_COMPACT_BYTES(realIntgBytes);
            int pos = NUM_TOTAL_DIG - (fracInts + NEEDELEMENTS(intg));
            if (compactPart) {
                if (wholeInts) {
                    aries_memcpy((char *)(values + (pos + 1)), compact->data + compactPart, realIntgBytes - compactPart);
                }
                aries_memcpy((char *)(values + pos), compact->data, compactPart);
            } else if (wholeInts) {
                aries_memcpy((char *)(values + pos), compact->data, realIntgBytes);
            }
            if (signPos == INTG_PART) {
                char *temp = ((char *)(values + (INDEX_LAST_DIG - fracInts)));
                if (compactPart == realIntgBytes) {
                    // <= 3 bytes only
                    temp += compactPart - 1;
                } else {
                    // >=4 bytes, have one sort
                    temp += 3;
                }
                sign = GET_SIGN_FROM_BIT(*temp);
                *temp = *temp & 0x7f;
            }
        }
        if (signPos == ADDITIONAL_PART) {
            sign = compact->data[realFracBytes + realIntgBytes];
        }
        if (sign) {
            Negate();
        }
    }

    ARIES_HOST_DEVICE_NO_INLINE Decimal::Decimal(const char s[]) {
        initialize(0, 0, 0);
        bool success = StringToDecimal((char *) s);
        if (!success) {
            SET_ERR(error, ERR_STR_2_DEC);
        }
    }

    ARIES_HOST_DEVICE_NO_INLINE Decimal::Decimal( const char* s, int len )
    {
        initialize(0, 0, 0);
        bool success = StringToDecimal((char *) s, len );
        if (!success) {
            SET_ERR(error, ERR_STR_2_DEC);
        }
    }

    ARIES_HOST_DEVICE_NO_INLINE Decimal::Decimal(int8_t v) {
        initialize(TINYINT_PRECISION, DEFAULT_SCALE, ARIES_MODE_EMPTY);
        values[INDEX_LAST_DIG] = v;
    }

    ARIES_HOST_DEVICE_NO_INLINE Decimal::Decimal(int16_t v) {
        initialize(SMALLINT_PRECISION, DEFAULT_SCALE, ARIES_MODE_EMPTY);
        values[INDEX_LAST_DIG] = v;
    }

    ARIES_HOST_DEVICE_NO_INLINE Decimal::Decimal(int32_t v) {
        initialize(INT_PRECISION, DEFAULT_SCALE, ARIES_MODE_EMPTY);
        values[INDEX_LAST_DIG - 1] = v / PER_DEC_MAX_SCALE;
        values[INDEX_LAST_DIG] = v % PER_DEC_MAX_SCALE;
    }

    ARIES_HOST_DEVICE_NO_INLINE Decimal::Decimal(int64_t v) {
        initialize(BIGINT_PRECISION, DEFAULT_SCALE, ARIES_MODE_EMPTY);
        int64_t t = v / PER_DEC_MAX_SCALE;
        values[INDEX_LAST_DIG - 2] = t / PER_DEC_MAX_SCALE;
        values[INDEX_LAST_DIG - 1] = t % PER_DEC_MAX_SCALE;
        values[INDEX_LAST_DIG] = v % PER_DEC_MAX_SCALE;
    }

    ARIES_HOST_DEVICE_NO_INLINE Decimal::Decimal(uint8_t v) {
        initialize(TINYINT_PRECISION, DEFAULT_SCALE, ARIES_MODE_EMPTY);
        values[INDEX_LAST_DIG] = v;
    }

    ARIES_HOST_DEVICE_NO_INLINE Decimal::Decimal(uint16_t v) {
        initialize(SMALLINT_PRECISION, DEFAULT_SCALE, ARIES_MODE_EMPTY);
        values[INDEX_LAST_DIG] = v;
    }

    ARIES_HOST_DEVICE_NO_INLINE Decimal::Decimal(uint32_t v) {
        initialize(INT_PRECISION, DEFAULT_SCALE, ARIES_MODE_EMPTY);
        values[INDEX_LAST_DIG - 1] = v / PER_DEC_MAX_SCALE;
        values[INDEX_LAST_DIG] = v % PER_DEC_MAX_SCALE;
    }

    ARIES_HOST_DEVICE_NO_INLINE Decimal::Decimal(uint64_t v) {
        initialize(BIGINT_PRECISION, DEFAULT_SCALE, ARIES_MODE_EMPTY);
        int64_t t = v / PER_DEC_MAX_SCALE;
        values[INDEX_LAST_DIG - 2] = t / PER_DEC_MAX_SCALE;
        values[INDEX_LAST_DIG - 1] = t % PER_DEC_MAX_SCALE;
        values[INDEX_LAST_DIG] = v % PER_DEC_MAX_SCALE;
    }

    ARIES_HOST_DEVICE_NO_INLINE bool Decimal::ToCompactDecimal(char * buf, int len) {
        SignPos signPos;
        int fracBits = GetDecimalNeedBits(frac);
        int intgBits = GetDecimalNeedBits(intg);
        int compactFracBytes = NEEDBYTES(fracBits);
        int compactIntgBytes = NEEDBYTES(intgBits);
        if (HAS_FREE_BIT(intgBits)) {
            signPos = INTG_PART;
        } else if (HAS_FREE_BIT(fracBits)) {
            signPos = FRAC_PART;
        } else {
            signPos = ADDITIONAL_PART;
        }
        if (len != compactFracBytes + compactIntgBytes + (signPos == ADDITIONAL_PART)) {
            return false;
        }
        int sign = 0;
        if (isLessZero()) {
            sign = 1;
            Negate();
        }
        //handle Frac part
        int usedInts = NEEDELEMENTS(frac);
        if (compactFracBytes) {
            int compactPart = GET_COMPACT_BYTES(compactFracBytes);
            if (compactFracBytes != compactPart) {
                aries_memcpy(buf + compactIntgBytes, (char *)(values + (NUM_TOTAL_DIG - usedInts)), compactFracBytes - compactPart);
            }
            if (compactPart) {
                int v = values[INDEX_LAST_DIG] / GetPowers10(DIG_PER_INT32 - frac % DIG_PER_INT32);
                aries_memcpy(buf + (compactIntgBytes + compactFracBytes - compactPart), (char *)&v, compactPart);
            }
            if (signPos == FRAC_PART) {
                int signBytePos = compactIntgBytes + compactFracBytes - 1;
                //has at last one Integer, use last byte of last one Integer
                if (compactFracBytes != compactPart) {
                    signBytePos -= compactPart;
                }
                assert((buf[signBytePos] & 0x80) == 0x0);
                SET_SIGN_BIT(buf[signBytePos], sign);
            }
        }
        //handle Intg part
        if (compactIntgBytes) {
            usedInts += NEEDELEMENTS(intg); //used to indicating total used Ints
            int wholeInts = GET_WHOLE_INTS(compactIntgBytes);
            int compactPart = GET_COMPACT_BYTES(compactIntgBytes);
            if (compactPart) {
                if (wholeInts) {
                    aries_memcpy(buf + compactPart, (char *)(values + (NUM_TOTAL_DIG - usedInts + 1)), compactIntgBytes - compactPart);
                }
                aries_memcpy(buf, (char *)(values + (NUM_TOTAL_DIG - usedInts)), compactPart);
            } else if (wholeInts) {
                aries_memcpy(buf, (char *)(values + (NUM_TOTAL_DIG - usedInts)), compactIntgBytes);
            }
            if (signPos == INTG_PART) {
                //sign bit is in last byte of intg part
                assert((buf[compactIntgBytes - 1] & 0x80) == 0x0);
                SET_SIGN_BIT(buf[compactIntgBytes - 1], sign);
            }
        }
        if (signPos == ADDITIONAL_PART) {
            buf[compactFracBytes + compactIntgBytes] = (char)sign;
        }

        if (sign) {
            Negate();
        }
        return true;
    }

    ARIES_HOST_DEVICE_NO_INLINE char *Decimal::GetInnerPrecisionScale(char result[]) {
        char temp[8];
        aries_sprintf(temp, "%d", intg + frac);
        aries_strcpy(result, temp);
        aries_strcat(result, ",");
        aries_sprintf((char *) temp, "%d", frac);
        aries_strcat(result, temp);
        return result;
    }

    ARIES_HOST_DEVICE_NO_INLINE char *Decimal::GetTargetPrecisionScale(char result[]) {
        return GetInnerPrecisionScale(result);
    }

    ARIES_HOST_DEVICE_NO_INLINE char *Decimal::GetPrecisionScale(char result[]) {
        if (GET_CALC_INTG(mode) + GET_CALC_FRAC(error) == 0) {
            return GetInnerPrecisionScale(result);
        }
        char temp[8];
        aries_sprintf(temp, "%d", GET_CALC_INTG(mode) + GET_CALC_FRAC(error));
        aries_strcpy(result, temp);
        aries_strcat(result, ",");
        aries_sprintf((char *) temp, "%d", GET_CALC_FRAC(error));
        aries_strcat(result, temp);
        return result;
    }

    ARIES_HOST_DEVICE_NO_INLINE uint16_t Decimal::GetSqlMode() {
        return GET_MODE(mode);
    }

    ARIES_HOST_DEVICE_NO_INLINE uint16_t Decimal::GetError() {
        return GET_ERR(error);
    }

    ARIES_HOST_DEVICE_NO_INLINE char *Decimal::GetInnerDecimal(char result[]) {
        char temp[16];
        int frac0 = NEEDELEMENTS(frac);
        //check sign
        bool postive = true;
        #pragma unroll
        for (int i = 0; i < NUM_TOTAL_DIG; i++) {
            if (values[i] < 0) {
                postive = false;
                break;
            }
        }
        //handle integer part
        int start = -1;
        int end = NUM_TOTAL_DIG - frac0;
        for (int i = 0; i < end; i++) {
            if (values[i] == 0)
                continue;
            start = i;
            break;
        }
        if (start == -1) {
            aries_strcpy(result, postive ? "0" : "-0");
        } else {
            aries_sprintf(result, "%d", values[start++]);
            for (int i = start; i < NUM_TOTAL_DIG - frac0; i++) {
                aries_sprintf(temp, values[i] < 0 ? "%010d" : "%09d", values[i]);
                aries_strcat(result, values[i] < 0 ? temp + 1 : temp);
            }
        }
        //handle frac part
        if (frac0) {
            aries_strcat(result, ".");
            int start = NUM_TOTAL_DIG - frac0;
            for ( int i = start; i < start + frac / DIG_PER_INT32; i++) {
                aries_sprintf(temp, values[i] < 0 ? "%010d" : "%09d", values[i]);
                aries_strcat(result, values[i] < 0 ? temp + 1 : temp);
            }
            //handle last one
            int remainLen = frac % DIG_PER_INT32;
            if (remainLen) {
                aries_sprintf(temp, values[INDEX_LAST_DIG] < 0 ? "%010d" : "%09d", values[INDEX_LAST_DIG]);
                aries_strncat(result, values[INDEX_LAST_DIG] < 0 ? temp + 1 : temp, remainLen);
            }
        }
        return result;
    }

    ARIES_HOST_DEVICE_NO_INLINE char * Decimal::GetDecimal(char result[]) {
        int frac0 = GET_CALC_FRAC(error), intg0 = GET_CALC_INTG(mode);
        if (frac0 == 0 && intg0 == 0) {
            return GetInnerDecimal(result);
        }
        if (frac0 != frac || intg0 != intg) {
            //need cast
            Decimal tmp(GET_CALC_INTG(mode) + GET_CALC_FRAC(error), GET_CALC_FRAC(error), GET_MODE(mode));
            SET_ERR(tmp.error, GET_ERR(error));
            tmp.cast(*this);
            return tmp.GetInnerDecimal(result);
        }
        return GetInnerDecimal(result);
    }

    ARIES_HOST_DEVICE_NO_INLINE void Decimal::CheckOverFlow() {
        int intg0 = intg == 0 ? 0 : NEEDELEMENTS(intg);
        int frac0 = frac == 0 ? 0 : NEEDELEMENTS(frac);
        int hiScale = intg0 * DIG_PER_INT32 - intg;
        bool neg = *this < 0;
        if (neg) {
            Negate();
        }
        //cross over values
        if (hiScale == 0) {
            intg0 += 1;
        } else {
            hiScale = DIG_PER_INT32 - hiScale;
        }
        int32_t hiMax = GetPowers10(hiScale) - 1;
        int st = NUM_TOTAL_DIG - frac0 - intg0;
        //check highest value
        int over = values[st] > hiMax ? 1 : 0;
        if (!over) {
            for (int i = 0; i < st; ++i) {
                if (values[i]) {
                    over = 1;
                    break;
                }
            }
        }
        if (over) {
            if (GET_MODE(mode) == ARIES_MODE_STRICT_ALL_TABLES) {
                SET_ERR(error, ERR_OVER_FLOW);
            }
            GenMaxDecByPrecision();
        }
        if (neg) {
            Negate();
        }
    }

    /*
     * integer/frac part by pos index
     *   0: value of 0 int
     *   1: value of 1 int
     *   2: value of 2 int
     *   3: value of 3 int
     * */
    ARIES_HOST_DEVICE_NO_INLINE void Decimal::setIntPart(int value, int pos) {
        int frac0 = NEEDELEMENTS(frac);
        int set = frac0 + pos;
        if (set < NUM_TOTAL_DIG) {
            values[INDEX_LAST_DIG - set] = value;
        }
    }

    ARIES_HOST_DEVICE_NO_INLINE void Decimal::setFracPart(int value, int pos) {
        int frac0 = NEEDELEMENTS(frac);
        if (pos < frac0) {
            values[INDEX_LAST_DIG - pos] = value;
        }
    }

    ARIES_HOST_DEVICE_NO_INLINE int Decimal::getIntPart(int pos) const {
        int frac0 = NEEDELEMENTS(frac);
        int get = frac0 + pos;
        if (get >= NUM_TOTAL_DIG) {
            return 0;
        }
        return values[INDEX_LAST_DIG - get];
    }

    ARIES_HOST_DEVICE_NO_INLINE int Decimal::getFracPart(int pos) const {
        int frac0 = NEEDELEMENTS(frac);
        if (pos >= frac0) {
            return 0;
        }
        return values[INDEX_LAST_DIG - pos];
    }

    //global method
    ARIES_HOST_DEVICE_NO_INLINE Decimal abs(Decimal decimal) {
        #pragma unroll
        for (int i = 0; i < NUM_TOTAL_DIG; i++) {
            if (decimal.values[i] < 0) {
                decimal.values[i] = -decimal.values[i];
            }
        }
        return decimal;
    }

    ARIES_HOST_DEVICE_NO_INLINE int GetDecimalRealBytes(uint16_t precision, uint16_t scale) {
        int fracBits = GetDecimalNeedBits(scale);
        int intgBits = GetDecimalNeedBits(precision - scale);
        if (HAS_FREE_BIT(fracBits) || HAS_FREE_BIT(intgBits)) {
            return NEEDBYTES(fracBits) +  NEEDBYTES(intgBits);
        } else {
            return NEEDBYTES(fracBits) +  NEEDBYTES(intgBits) + 1;
        }
    }

    ARIES_HOST_DEVICE_NO_INLINE int GetDecimalNeedBits(int base10Precision) {
        int len = base10Precision / DIG_PER_INT32 * 32;
        switch (base10Precision % DIG_PER_INT32) {
            case 0:
                len += 0;
                break;
            case 1:
                len += 4;
                break;
            case 2:
                len += 7;
                break;
            case 3:
                len += 10;
                break;
            case 4:
                len += 14;
                break;
            case 5:
                len += 17;
                break;
            case 6:
                len += 20;
                break;
            case 7:
                len += 24;
                break;
            case 8:
                len += 27;
                break;
        }
        return len;
    }

    ARIES_HOST_DEVICE_NO_INLINE int GetDecimalValidElementsCount( uint16_t precision, uint16_t scale )
    {
        return NEEDELEMENTS( precision - scale ) + NEEDELEMENTS( scale );
    }

    ARIES_HOST_DEVICE_NO_INLINE Decimal& Decimal::cast(const Decimal &v) {
        if (frac >= v.frac) {
            SET_ERR(error, GET_ERR(v.error));
            int shift = NEEDELEMENTS(frac) - NEEDELEMENTS(v.frac);
            for (int i = 0; i < shift; ++i) {
                values[i] = 0;
            }
            for (int i = shift; i < NUM_TOTAL_DIG; ++i) {
                values[i - shift] = v.values[i];
            }
            if (intg < v.intg) {
                CheckOverFlow();
            }
        } else {
            if (!v.isFracZero()) {
                int shift = NEEDELEMENTS(v.frac) - NEEDELEMENTS(frac);
                for (int i = 0; i < shift; ++i) {
                    values[i] = 0;
                }
                for (int i = shift; i < NUM_TOTAL_DIG; ++i) {
                    values[i] = v.values[i - shift];
                }
                bool neg = *this < 0;
                if (neg) {
                    Negate();
                }
                //scale down
                int scale = frac;
                if ( scale >= DIG_PER_INT32) {
                    scale %= DIG_PER_INT32;
                }
                if (scale) {
                    // scale 5: 123456789 -> 123460000
                    values[INDEX_LAST_DIG] = values[INDEX_LAST_DIG] / GetPowers10( DIG_PER_INT32 - scale) * GetPowers10( DIG_PER_INT32 - scale);
                }

                //check the carry if cast
                //scale 9, check 1 of next value
                if (++scale == 1) {
                    //use shift as index of values later, change check frac value index
                    --shift;
                }
                scale = DIG_PER_INT32 - scale;
                if (aries_abs(v.values[INDEX_LAST_DIG - shift] / GetPowers10(scale)) % 10 >= 5) {
                    int max = GetPowers10( DIG_PER_INT32);
                    int carry = scale + 1 == DIG_PER_INT32 ? 1 : GetPowers10( scale + 1);
                    for (int i = INDEX_LAST_DIG; i >= 0; --i) {
                        values[i] += carry;
                        if (values[i] < max) {
                            carry = 0;
                            break;
                        }
                        carry = 1;
                        values[i] = 0;
                    }
                    // check highest one
                    if (carry == 1) {
                        values[0] = max;
                    }
                }
                if (neg) {
                    Negate();
                }
            }
            CheckOverFlow();
        }
        assert(intg + frac <= SUPPORTED_MAX_PRECISION && frac <= SUPPORTED_MAX_SCALE);
        return *this;
    }

    ARIES_HOST_DEVICE_NO_INLINE Decimal& Decimal::truncate( int p ) {
        uint16_t frac0 = frac, intg0 = intg;
        CalcInnerTruncatePrecision(p);
        CalcTruncatePrecision(p);
        if (p > 0) {
            p = frac;
        } else {
            if (-p >= intg0) {
                //result should be zero
                p = -(NEEDELEMENTS(intg0) + NEEDELEMENTS(frac0)) * DIG_PER_INT32;
            }
        }
        int shift = p >= 0 ? NEEDELEMENTS(frac0) - NEEDELEMENTS(p) : NEEDELEMENTS(frac0);
        if (shift > 0) {
            for ( int i = INDEX_LAST_DIG - shift; i >= 0; --i ) {
                values[i + shift] = values[i];
            }
            for ( int i = 0; i < shift; ++i )
            {
                values[i] = 0;
            }
        } else if (shift < 0) {
            for ( int i = -shift; i < NUM_TOTAL_DIG; ++i ) {
                values[i + shift] = values[i];
            }
            for ( int i = NUM_TOTAL_DIG + shift; i < NUM_TOTAL_DIG; ++i )
            {
                values[i] = 0;
            }
        }
        if (frac0 > p) {
            int cutPowersN = p > 0 ? (DIG_PER_INT32 - p) % DIG_PER_INT32 : -p;
            int cutInt = cutPowersN / DIG_PER_INT32;
            int cutPowers10 = cutPowersN % DIG_PER_INT32;
            if (cutInt) {
                int cutStartIndex = INDEX_LAST_DIG - (cutPowers10 ? 1 : 0);
                for (int i = cutStartIndex; i > cutStartIndex - cutInt; --i) {
                    values[i] = 0;
                }
            }
            if (cutPowers10) {
                values[INDEX_LAST_DIG] -= values[INDEX_LAST_DIG] % GetPowers10(cutPowers10);
            }
        }
        return *this;
    }

    ARIES_HOST_DEVICE_NO_INLINE void Decimal::CalcTruncTargetPrecision( int p ) {
        frac = p >= 0 ? aries_min(p, SUPPORTED_MAX_SCALE) : 0;
        if (-p >= intg) {
            intg = 1;
        }
    }

    ARIES_HOST_DEVICE_NO_INLINE void Decimal::CalcTruncatePrecision( int p ) {
        if (GET_CALC_INTG(mode) == 0 && GET_CALC_FRAC(error) == 0) {
            SET_CALC_INTG(mode, intg);
            SET_CALC_FRAC(error, frac);
        }
        uint16_t frac0 = p >= 0 ? aries_min(p, SUPPORTED_MAX_SCALE) : 0;
        uint16_t intg0 = GET_CALC_INTG(mode);
        if (-p >= intg0) {
            intg0 = 1;
        }
        uint8_t e = 0;
        FIX_INTG_FRAC_ERROR(SUPPORTED_MAX_PRECISION, intg0, frac0, e);
        SET_CALC_INTG(mode, intg0);
        SET_CALC_FRAC(error,frac0);
        SET_ERR(error, e);
    }

    ARIES_HOST_DEVICE_NO_INLINE void Decimal::CalcInnerTruncatePrecision( int p ) {
        uint16_t frac0 = p >= 0 ? aries_min(p, SUPPORTED_MAX_SCALE) : 0;
        uint16_t intg0 = intg;
        if (-p >= intg) {
            intg0 = 1;
        }
        uint16_t frac1, frac2;
        frac1 = frac2 = NEEDELEMENTS(frac0);
        uint16_t intg1, intg2;
        intg1 = intg2 = NEEDELEMENTS(intg0);
        uint8_t e = 0;
        FIX_INTG_FRAC_ERROR(INNER_MAX_PRECISION_INT32_NUM, intg1, frac1, e);
        SET_PREC_SCALE_VALUE(frac, frac0, frac1, frac2);
        SET_PREC_SCALE_VALUE(intg, intg0, intg1, intg2);
        SET_ERR(error, e);
    }

    ARIES_HOST_DEVICE_NO_INLINE Decimal::operator bool() const {
        return !isZero();
    }

    ARIES_HOST_DEVICE_NO_INLINE Decimal Decimal::operator-() {
        Decimal decimal(*this);
        #pragma unroll
        for (int i = 0; i < NUM_TOTAL_DIG; i++) {
            decimal.values[i] = -decimal.values[i];
        }
        return decimal;
    }

    //signed
    ARIES_HOST_DEVICE_NO_INLINE Decimal &Decimal::operator=(int8_t v) {
        Decimal tmp(v);
        SET_MODE(tmp.mode, GET_MODE(mode));
        *this = tmp;
        return *this;
    }

    ARIES_HOST_DEVICE_NO_INLINE Decimal &Decimal::operator=(int16_t v) {
        Decimal tmp(v);
        SET_MODE(tmp.mode, GET_MODE(mode));
        *this = tmp;
        return *this;
    }

    ARIES_HOST_DEVICE_NO_INLINE Decimal &Decimal::operator=(int32_t v) {
        Decimal tmp(v);
        SET_MODE(tmp.mode, GET_MODE(mode));
        *this = tmp;
        return *this;
    }

    ARIES_HOST_DEVICE_NO_INLINE Decimal &Decimal::operator=(int64_t v) {
        Decimal tmp(v);
        SET_MODE(tmp.mode, GET_MODE(mode));
        *this = tmp;
        return *this;
    }

    //unsigned
    ARIES_HOST_DEVICE_NO_INLINE Decimal &Decimal::operator=(uint8_t v) {
        Decimal tmp(v);
        SET_MODE(tmp.mode, GET_MODE(mode));
        *this = tmp;
        return *this;
    }

    ARIES_HOST_DEVICE_NO_INLINE Decimal &Decimal::operator=(uint16_t v) {
        Decimal tmp(v);
        SET_MODE(tmp.mode, GET_MODE(mode));
        *this = tmp;
        return *this;
    }

    ARIES_HOST_DEVICE_NO_INLINE Decimal &Decimal::operator=(uint32_t v) {
        Decimal tmp(v);
        SET_MODE(tmp.mode, GET_MODE(mode));
        *this = tmp;
        return *this;
    }

    ARIES_HOST_DEVICE_NO_INLINE Decimal &Decimal::operator=(uint64_t v) {
        Decimal tmp(v);
        SET_MODE(tmp.mode, GET_MODE(mode));
        *this = tmp;
        return *this;
    }

    //for decimal
    ARIES_HOST_DEVICE_NO_INLINE bool operator>(const Decimal &left, const Decimal &right) {
        int temp;
        if (ALIGNED(left.frac, right.frac)) {
            #pragma unroll
            for (int i = 0; i < NUM_TOTAL_DIG; i++) {
                if ((temp = (left.values[i] - right.values[i]))) {
                    return temp > 0;
                }
            }
        } else {
            Decimal l(left);
            Decimal r(right);
            l.AlignAddSubData(r);
            #pragma unroll
            for (int i = 0; i < NUM_TOTAL_DIG; i++) {
                if ((temp = (l.values[i] - r.values[i]))) {
                    return temp > 0;
                }
            }
        }
        return false;
    }

    ARIES_HOST_DEVICE_NO_INLINE bool operator>=(const Decimal &left, const Decimal &right) {
        return !(left < right);
    }

    ARIES_HOST_DEVICE_NO_INLINE bool operator<(const Decimal &left, const Decimal &right) {
        int temp;
        if (ALIGNED(left.frac, right.frac)) {
            #pragma unroll
            for (int i = 0; i < NUM_TOTAL_DIG; i++) {
                if ((temp = (left.values[i] - right.values[i]))) {
                    return temp < 0;
                }
            }
        } else {
            Decimal l(left);
            Decimal r(right);
            l.AlignAddSubData(r);
            #pragma unroll
            for (int i = 0; i < NUM_TOTAL_DIG; i++) {
                if ((temp = (l.values[i] - r.values[i]))) {
                    return temp < 0;
                }
            }
        }
        return false;
    }

    ARIES_HOST_DEVICE_NO_INLINE bool operator<=(const Decimal &left, const Decimal &right) {
        return !(left > right);
    }

    ARIES_HOST_DEVICE_NO_INLINE bool operator==(const Decimal &left, const Decimal &right) {
        if (ALIGNED(left.frac, right.frac)) {
            for (int i = 0; i < NUM_TOTAL_DIG; i++) {
                if (left.values[i] - right.values[i]) {
                    return false;
                }
            }
        } else {
            Decimal l(left);
            Decimal r(right);
            l.AlignAddSubData(r);
            #pragma unroll
            for (int i = 0; i < NUM_TOTAL_DIG; i++) {
                if (l.values[i] - r.values[i]) {
                    return false;
                }
            }
        }
        return true;
    }

    ARIES_HOST_DEVICE_NO_INLINE bool operator!=(const Decimal &left, const Decimal &right) {
        return !(left == right);
    }

    // for int8_t
    ARIES_HOST_DEVICE_NO_INLINE bool operator>(int8_t left, const Decimal &right) {
        return (int32_t) left > right;
    }

    ARIES_HOST_DEVICE_NO_INLINE bool operator>=(int8_t left, const Decimal &right) {
        return (int32_t) left >= right;
    }

    ARIES_HOST_DEVICE_NO_INLINE bool operator<(int8_t left, const Decimal &right) {
        return (int32_t) left < right;
    }

    ARIES_HOST_DEVICE_NO_INLINE bool operator<=(int8_t left, const Decimal &right) {
        return (int32_t) left <= right;
    }

    ARIES_HOST_DEVICE_NO_INLINE bool operator==(int8_t left, const Decimal &right) {
        return (int32_t) left == right;
    }

    ARIES_HOST_DEVICE_NO_INLINE bool operator!=(int8_t left, const Decimal &right) {
        return !(left == right);
    }

    ARIES_HOST_DEVICE_NO_INLINE bool operator>(const Decimal &left, int8_t right) {
        return left > (int32_t) right;
    }

    ARIES_HOST_DEVICE_NO_INLINE bool operator>=(const Decimal &left, int8_t right) {
        return left >= (int32_t) right;
    }

    ARIES_HOST_DEVICE_NO_INLINE bool operator<(const Decimal &left, int8_t right) {
        return left < (int32_t) right;
    }

    ARIES_HOST_DEVICE_NO_INLINE bool operator<=(const Decimal &left, int8_t right) {
        return left <= (int32_t) right;
    }

    ARIES_HOST_DEVICE_NO_INLINE bool operator==(const Decimal &left, int8_t right) {
        return left == (int32_t) right;
    }

    ARIES_HOST_DEVICE_NO_INLINE bool operator!=(const Decimal &left, int8_t right) {
        return left != (int32_t) right;
    }

    // for uint8_t
    ARIES_HOST_DEVICE_NO_INLINE bool operator>(uint8_t left, const Decimal &right) {
        return (uint32_t) left > right;
    }

    ARIES_HOST_DEVICE_NO_INLINE bool operator>=(uint8_t left, const Decimal &right) {
        return (uint32_t) left >= right;
    }

    ARIES_HOST_DEVICE_NO_INLINE bool operator<(uint8_t left, const Decimal &right) {
        return (uint32_t) left < right;
    }

    ARIES_HOST_DEVICE_NO_INLINE bool operator<=(uint8_t left, const Decimal &right) {
        return (uint32_t) left <= right;
    }

    ARIES_HOST_DEVICE_NO_INLINE bool operator==(uint8_t left, const Decimal &right) {
        return (uint32_t) left == right;
    }

    ARIES_HOST_DEVICE_NO_INLINE bool operator!=(uint8_t left, const Decimal &right) {
        return !(left == right);
    }

    ARIES_HOST_DEVICE_NO_INLINE bool operator>(const Decimal &left, uint8_t right) {
        return left > (uint32_t) right;
    }

    ARIES_HOST_DEVICE_NO_INLINE bool operator>=(const Decimal &left, uint8_t right) {
        return left >= (uint32_t) right;
    }

    ARIES_HOST_DEVICE_NO_INLINE bool operator<(const Decimal &left, uint8_t right) {
        return left < (uint32_t) right;
    }

    ARIES_HOST_DEVICE_NO_INLINE bool operator<=(const Decimal &left, uint8_t right) {
        return left <= (uint32_t) right;
    }

    ARIES_HOST_DEVICE_NO_INLINE bool operator==(const Decimal &left, uint8_t right) {
        return left == (uint32_t) right;
    }

    ARIES_HOST_DEVICE_NO_INLINE bool operator!=(const Decimal &left, uint8_t right) {
        return left != (uint32_t) right;
    }

    //for int16_t
    ARIES_HOST_DEVICE_NO_INLINE bool operator>(int16_t left, const Decimal &right) {
        return (int32_t) left > right;
    }

    ARIES_HOST_DEVICE_NO_INLINE bool operator>=(int16_t left, const Decimal &right) {
        return (int32_t) left >= right;
    }

    ARIES_HOST_DEVICE_NO_INLINE bool operator<(int16_t left, const Decimal &right) {
        return (int32_t) left < right;
    }

    ARIES_HOST_DEVICE_NO_INLINE bool operator<=(int16_t left, const Decimal &right) {
        return (int32_t) left <= right;
    }

    ARIES_HOST_DEVICE_NO_INLINE bool operator==(int16_t left, const Decimal &right) {
        return (int32_t) left == right;
    }

    ARIES_HOST_DEVICE_NO_INLINE bool operator!=(int16_t left, const Decimal &right) {
        return (int32_t) left != right;
    }

    ARIES_HOST_DEVICE_NO_INLINE bool operator>(const Decimal &left, int16_t right) {
        return left > (int32_t) right;
    }

    ARIES_HOST_DEVICE_NO_INLINE bool operator>=(const Decimal &left, int16_t right) {
        return left >= (int32_t) right;
    }

    ARIES_HOST_DEVICE_NO_INLINE bool operator<(const Decimal &left, int16_t right) {
        return left < (int32_t) right;
    }

    ARIES_HOST_DEVICE_NO_INLINE bool operator<=(const Decimal &left, int16_t right) {
        return left <= (int32_t) right;
    }

    ARIES_HOST_DEVICE_NO_INLINE bool operator==(const Decimal &left, int16_t right) {
        return left == (int32_t) right;
    }

    ARIES_HOST_DEVICE_NO_INLINE bool operator!=(const Decimal &left, int16_t right) {
        return left != (int32_t) right;
    }

    //for uint16_t
    ARIES_HOST_DEVICE_NO_INLINE bool operator>(uint16_t left, const Decimal &right) {
        return (uint32_t) left > right;
    }

    ARIES_HOST_DEVICE_NO_INLINE bool operator>=(uint16_t left, const Decimal &right) {
        return (uint32_t) left >= right;
    }

    ARIES_HOST_DEVICE_NO_INLINE bool operator<(uint16_t left, const Decimal &right) {
        return (uint32_t) left < right;
    }

    ARIES_HOST_DEVICE_NO_INLINE bool operator<=(uint16_t left, const Decimal &right) {
        return (uint32_t) left <= right;
    }

    ARIES_HOST_DEVICE_NO_INLINE bool operator==(uint16_t left, const Decimal &right) {
        return (uint32_t) left == right;
    }

    ARIES_HOST_DEVICE_NO_INLINE bool operator!=(uint16_t left, const Decimal &right) {
        return (uint32_t) left != right;
    }

    ARIES_HOST_DEVICE_NO_INLINE bool operator>(const Decimal &left, uint16_t right) {
        return left > (uint32_t) right;
    }

    ARIES_HOST_DEVICE_NO_INLINE bool operator>=(const Decimal &left, uint16_t right) {
        return left >= (uint32_t) right;
    }

    ARIES_HOST_DEVICE_NO_INLINE bool operator<(const Decimal &left, uint16_t right) {
        return left < (uint32_t) right;
    }

    ARIES_HOST_DEVICE_NO_INLINE bool operator<=(const Decimal &left, uint16_t right) {
        return left <= (uint32_t) right;
    }

    ARIES_HOST_DEVICE_NO_INLINE bool operator==(const Decimal &left, uint16_t right) {
        return left == (uint32_t) right;
    }

    ARIES_HOST_DEVICE_NO_INLINE bool operator!=(const Decimal &left, uint16_t right) {
        return left != (uint32_t) right;
    }

    //for int32_t
    ARIES_HOST_DEVICE_NO_INLINE bool operator>(int32_t left, const Decimal &right) {
        Decimal d(left);
        return d > right;
    }

    ARIES_HOST_DEVICE_NO_INLINE bool operator>=(int32_t left, const Decimal &right) {
        Decimal d(left);
        return d >= right;
    }

    ARIES_HOST_DEVICE_NO_INLINE bool operator<(int32_t left, const Decimal &right) {
        Decimal d(left);
        return d < right;
    }

    ARIES_HOST_DEVICE_NO_INLINE bool operator<=(int32_t left, const Decimal &right) {
        Decimal d(left);
        return d <= right;
    }

    ARIES_HOST_DEVICE_NO_INLINE bool operator==(int32_t left, const Decimal &right) {
        Decimal d(left);
        return d == right;
    }

    ARIES_HOST_DEVICE_NO_INLINE bool operator!=(int32_t left, const Decimal &right) {
        Decimal d(left);
        return d != right;
    }

    ARIES_HOST_DEVICE_NO_INLINE bool operator>(const Decimal &left, int32_t right) {
        Decimal d(right);
        return left > d;
    }

    ARIES_HOST_DEVICE_NO_INLINE bool operator>=(const Decimal &left, int32_t right) {
        Decimal d(right);
        return left >= d;
    }

    ARIES_HOST_DEVICE_NO_INLINE bool operator<(const Decimal &left, int32_t right) {
        Decimal d(right);
        return left < d;
    }

    ARIES_HOST_DEVICE_NO_INLINE bool operator<=(const Decimal &left, int32_t right) {
        Decimal d(right);
        return left <= d;
    }

    ARIES_HOST_DEVICE_NO_INLINE bool operator==(const Decimal &left, int32_t right) {
        Decimal d(right);
        return left == d;
    }

    ARIES_HOST_DEVICE_NO_INLINE bool operator!=(const Decimal &left, int32_t right) {
        Decimal d(right);
        return left != d;
    }

    //for uint32_t
    ARIES_HOST_DEVICE_NO_INLINE bool operator>(uint32_t left, const Decimal &right) {
        Decimal d(left);
        return d > right;
    }

    ARIES_HOST_DEVICE_NO_INLINE bool operator>=(uint32_t left, const Decimal &right) {
        Decimal d(left);
        return d >= right;
    }

    ARIES_HOST_DEVICE_NO_INLINE bool operator<(uint32_t left, const Decimal &right) {
        Decimal d(left);
        return d < right;
    }

    ARIES_HOST_DEVICE_NO_INLINE bool operator<=(uint32_t left, const Decimal &right) {
        Decimal d(left);
        return d <= right;
    }

    ARIES_HOST_DEVICE_NO_INLINE bool operator==(uint32_t left, const Decimal &right) {
        Decimal d(left);
        return d == right;
    }

    ARIES_HOST_DEVICE_NO_INLINE bool operator!=(uint32_t left, const Decimal &right) {
        Decimal d(left);
        return d != right;
    }

    ARIES_HOST_DEVICE_NO_INLINE bool operator>(const Decimal &left, uint32_t right) {
        Decimal d(right);
        return left > d;
    }

    ARIES_HOST_DEVICE_NO_INLINE bool operator>=(const Decimal &left, uint32_t right) {
        Decimal d(right);
        return left >= d;
    }

    ARIES_HOST_DEVICE_NO_INLINE bool operator<(const Decimal &left, uint32_t right) {
        Decimal d(right);
        return left < d;
    }

    ARIES_HOST_DEVICE_NO_INLINE bool operator<=(const Decimal &left, uint32_t right) {
        Decimal d(right);
        return left <= d;
    }

    ARIES_HOST_DEVICE_NO_INLINE bool operator==(const Decimal &left, uint32_t right) {
        Decimal d(right);
        return left == d;
    }

    ARIES_HOST_DEVICE_NO_INLINE bool operator!=(const Decimal &left, uint32_t right) {
        Decimal d(right);
        return left != d;
    }

    //for int64_t
    ARIES_HOST_DEVICE_NO_INLINE bool operator>(int64_t left, const Decimal &right) {
        Decimal d(left);
        return d > right;
    }

    ARIES_HOST_DEVICE_NO_INLINE bool operator>=(int64_t left, const Decimal &right) {
        Decimal d(left);
        return d >= right;
    }

    ARIES_HOST_DEVICE_NO_INLINE bool operator<(int64_t left, const Decimal &right) {
        Decimal d(left);
        return d < right;
    }

    ARIES_HOST_DEVICE_NO_INLINE bool operator<=(int64_t left, const Decimal &right) {
        Decimal d(left);
        return d <= right;
    }

    ARIES_HOST_DEVICE_NO_INLINE bool operator==(int64_t left, const Decimal &right) {
        Decimal d(left);
        return d == right;
    }

    ARIES_HOST_DEVICE_NO_INLINE bool operator!=(int64_t left, const Decimal &right) {
        Decimal d(left);
        return d != right;
    }

    ARIES_HOST_DEVICE_NO_INLINE bool operator>(const Decimal &left, int64_t right) {
        Decimal d(right);
        return left > d;
    }

    ARIES_HOST_DEVICE_NO_INLINE bool operator>=(const Decimal &left, int64_t right) {
        Decimal d(right);
        return left >= d;
    }

    ARIES_HOST_DEVICE_NO_INLINE bool operator<(const Decimal &left, int64_t right) {
        Decimal d(right);
        return left < d;
    }

    ARIES_HOST_DEVICE_NO_INLINE bool operator<=(const Decimal &left, int64_t right) {
        Decimal d(right);
        return left <= d;
    }

    ARIES_HOST_DEVICE_NO_INLINE bool operator==(const Decimal &left, int64_t right) {
        Decimal d(right);
        return left == d;
    }

    ARIES_HOST_DEVICE_NO_INLINE bool operator!=(const Decimal &left, int64_t right) {
        Decimal d(right);
        return left != d;
    }

    //for uint64_t
    ARIES_HOST_DEVICE_NO_INLINE bool operator>(uint64_t left, const Decimal &right) {
        Decimal d(left);
        return d > right;
    }

    ARIES_HOST_DEVICE_NO_INLINE bool operator>=(uint64_t left, const Decimal &right) {
        Decimal d(left);
        return d >= right;
    }

    ARIES_HOST_DEVICE_NO_INLINE bool operator<(uint64_t left, const Decimal &right) {
        Decimal d(left);
        return d < right;
    }

    ARIES_HOST_DEVICE_NO_INLINE bool operator<=(uint64_t left, const Decimal &right) {
        Decimal d(left);
        return d <= right;
    }

    ARIES_HOST_DEVICE_NO_INLINE bool operator==(uint64_t left, const Decimal &right) {
        Decimal d(left);
        return d == right;
    }

    ARIES_HOST_DEVICE_NO_INLINE bool operator!=(uint64_t left, const Decimal &right) {
        Decimal d(left);
        return d != right;
    }

    ARIES_HOST_DEVICE_NO_INLINE bool operator>(const Decimal &left, uint64_t right) {
        Decimal d(right);
        return left > d;
    }

    ARIES_HOST_DEVICE_NO_INLINE bool operator>=(const Decimal &left, uint64_t right) {
        Decimal d(right);
        return left >= d;
    }

    ARIES_HOST_DEVICE_NO_INLINE bool operator<(const Decimal &left, uint64_t right) {
        Decimal d(right);
        return left < d;
    }

    ARIES_HOST_DEVICE_NO_INLINE bool operator<=(const Decimal &left, uint64_t right) {
        Decimal d(right);
        return left <= d;
    }

    ARIES_HOST_DEVICE_NO_INLINE bool operator==(const Decimal &left, uint64_t right) {
        Decimal d(right);
        return left == d;
    }

    ARIES_HOST_DEVICE_NO_INLINE bool operator!=(const Decimal &left, uint64_t right) {
        Decimal d(right);
        return left != d;
    }

    //for float
    ARIES_HOST_DEVICE_NO_INLINE bool operator>(float left, const Decimal &right) {
        return (double) left > right;
    }

    ARIES_HOST_DEVICE_NO_INLINE bool operator>=(float left, const Decimal &right) {
        return (double) left >= right;
    }

    ARIES_HOST_DEVICE_NO_INLINE bool operator<(float left, const Decimal &right) {
        return (double) left < right;
    }

    ARIES_HOST_DEVICE_NO_INLINE bool operator<=(float left, const Decimal &right) {
        return (double) left <= right;
    }

    ARIES_HOST_DEVICE_NO_INLINE bool operator==(float left, const Decimal &right) {
        return (double) left == right;
    }

    ARIES_HOST_DEVICE_NO_INLINE bool operator!=(float left, const Decimal &right) {
        return (double) left != right;
    }

    ARIES_HOST_DEVICE_NO_INLINE bool operator>(const Decimal &left, float right) {
        return left > (double) right;
    }

    ARIES_HOST_DEVICE_NO_INLINE bool operator>=(const Decimal &left, float right) {
        return left >= (double) right;
    }

    ARIES_HOST_DEVICE_NO_INLINE bool operator<(const Decimal &left, float right) {
        return left < (double) right;
    }

    ARIES_HOST_DEVICE_NO_INLINE bool operator<=(const Decimal &left, float right) {
        return left <= (double) right;
    }

    ARIES_HOST_DEVICE_NO_INLINE bool operator==(const Decimal &left, float right) {
        return left == (double) right;
    }

    ARIES_HOST_DEVICE_NO_INLINE bool operator!=(const Decimal &left, float right) {
        return left != (double) right;
    }

    //for double
    ARIES_HOST_DEVICE_NO_INLINE bool operator>(double left, const Decimal &right) {
        return left > right.GetDouble();
    }

    ARIES_HOST_DEVICE_NO_INLINE bool operator>=(double left, const Decimal &right) {
        return left >= right.GetDouble();
    }

    ARIES_HOST_DEVICE_NO_INLINE bool operator<(double left, const Decimal &right) {
        return left < right.GetDouble();
    }

    ARIES_HOST_DEVICE_NO_INLINE bool operator<=(double left, const Decimal &right) {
        return left <= right.GetDouble();
    }

    ARIES_HOST_DEVICE_NO_INLINE bool operator==(double left, const Decimal &right) {
        return left == right.GetDouble();
    }

    ARIES_HOST_DEVICE_NO_INLINE bool operator!=(double left, const Decimal &right) {
        return left != right.GetDouble();
    }

    ARIES_HOST_DEVICE_NO_INLINE bool operator>(const Decimal &left, double right) {
        return left.GetDouble() > right;
    }

    ARIES_HOST_DEVICE_NO_INLINE bool operator>=(const Decimal &left, double right) {
        return left.GetDouble() >= right;
    }

    ARIES_HOST_DEVICE_NO_INLINE bool operator<(const Decimal &left, double right) {
        return left.GetDouble() < right;
    }

    ARIES_HOST_DEVICE_NO_INLINE bool operator<=(const Decimal &left, double right) {
        return left.GetDouble() <= right;
    }

    ARIES_HOST_DEVICE_NO_INLINE bool operator==(const Decimal &left, double right) {
        return left.GetDouble() == right;
    }

    ARIES_HOST_DEVICE_NO_INLINE bool operator!=(const Decimal &left, double right) {
        return left.GetDouble() != right;
    }

    // for add
    ARIES_HOST_DEVICE_NO_INLINE void Decimal::CalcInnerAddPrecision(const Decimal& d) {
        uint16_t frac0 = aries_min(aries_max(frac, d.frac), SUPPORTED_MAX_SCALE);
        uint16_t intg0 = aries_max(intg, d.intg);
        int highestV1, highestV2, i1 = GetRealIntgSize(highestV1), i2 = d.GetRealIntgSize(highestV2);
        if (aries_max(i1, i2) >= NEEDELEMENTS(intg0)) {
            int value = i1 > i2 ? highestV1 : i1 < i2 ? highestV2 : highestV1 + highestV2;
            int maxIntg = intg0 % DIG_PER_INT32;
            if (maxIntg == 0) {
                maxIntg = DIG_PER_INT32;
            }
            if (value && (aries_abs(value) >= GetPowers10(maxIntg) - 1)) {
                intg0 += 1;
            }
        }
        uint16_t frac1, frac2;
        frac1 = frac2 = NEEDELEMENTS(frac0);
        uint16_t intg1, intg2;
        intg1 = intg2 = NEEDELEMENTS(intg0);
        uint8_t e = 0;
        FIX_INTG_FRAC_ERROR(INNER_MAX_PRECISION_INT32_NUM, intg1, frac1, e);
        SET_PREC_SCALE_VALUE(frac, frac0, frac1, frac2);
        SET_PREC_SCALE_VALUE(intg, intg0, intg1, intg2);
        SET_ERR(error, e);
    }

    ARIES_HOST_DEVICE_NO_INLINE void Decimal::CalcAddPrecision(const Decimal &d) {
        uint16_t frac0 = aries_min(aries_max(GET_CALC_FRAC(error), GET_CALC_FRAC(d.error)), SUPPORTED_MAX_SCALE);
        uint16_t intg0 = aries_max(GET_CALC_INTG(mode), GET_CALC_INTG(d.mode));
        int highestV1, highestV2, i1 = GetRealIntgSize(highestV1), i2 = d.GetRealIntgSize(highestV2);
        if (aries_max(i1, i2) >= NEEDELEMENTS(intg0)) {
            int value = i1 > i2 ? highestV1 : i1 < i2 ? highestV2 : highestV1 + highestV2;
            int maxIntg = intg0 % DIG_PER_INT32;
            if (maxIntg == 0) {
                maxIntg = DIG_PER_INT32;
            }
            if (value && (aries_abs(value) >= GetPowers10(maxIntg) - 1)) {
                intg0 += 1;
            }
        }
        uint8_t e = 0;
        FIX_INTG_FRAC_ERROR(SUPPORTED_MAX_PRECISION, intg0, frac0, e);
        SET_CALC_INTG(mode, intg0);
        SET_CALC_FRAC(error,frac0);
        SET_ERR(error, e);
    }

    ARIES_HOST_DEVICE_NO_INLINE void Decimal::CalcAddTargetPrecision( const Decimal& d ) {
        uint16_t frac0 = aries_min(aries_max(frac, d.frac), SUPPORTED_MAX_SCALE);
        uint16_t intg0 = aries_max(intg, d.intg) + 1;
        uint8_t e = 0;
        FIX_TAGET_INTG_FRAC_ERROR(SUPPORTED_MAX_PRECISION, intg0, frac0, e);
        intg = intg0;
        frac = frac0;
        error = e;
    }

    ARIES_HOST_DEVICE_NO_INLINE Decimal &Decimal::AddBothPositiveNums(Decimal &d) {
        AlignAddSubData(d);
        //add
        int32_t carry = 0;
        for (int i = INDEX_LAST_DIG; i >= 0; i--) {
            values[i] += d.values[i];
            values[i] += carry;
            if (values[i] >= PER_DEC_MAX_SCALE) {
                carry = 1;
                values[i] -= PER_DEC_MAX_SCALE;
            } else {
                carry = 0;
            }
        }
        //        CheckOverFlow();
        return *this;
    }

    ARIES_HOST_DEVICE_NO_INLINE Decimal &Decimal::operator+=(const Decimal &d) {
        CheckAndSetCalcPrecision();
        Decimal added(d);
        added.CheckAndSetCalcPrecision();
        //calculate precision after plus
        uint8_t intg0, frac0, mode0, error0;
        if (1 > 0) {
            Decimal calcPrecision(*this);
            calcPrecision.CalcAddPrecision(added);
            calcPrecision.CalcInnerAddPrecision(added);
            intg0 = calcPrecision.intg;
            frac0 = calcPrecision.frac;
            mode0 = calcPrecision.mode;
            error0 = calcPrecision.error;
        }
        bool addedNeg = added.isLessZero();
        if (isLessZero())  //-
        {
            Negate();
            if (addedNeg)  // --
            {
                //-a + -b = - (a + b)
                added.Negate();
                AddBothPositiveNums(added);
            } else //-+
            {
                //-a + b = - (a - b)
                SubBothPositiveNums(added);
            }
            Negate();
        } else {
            if (addedNeg) //+ -
            {
                // a + -b = a - (-b)
                added.Negate();
                SubBothPositiveNums(added);
            } else {
                AddBothPositiveNums(added);
            }
        }
        //set precision
        intg = intg0;
        frac = frac0;
        mode = mode0;
        error = error0;
        return *this;
    }

    //signed
    ARIES_HOST_DEVICE_NO_INLINE Decimal &Decimal::operator+=(int8_t i) {
        Decimal d(i);
        return *this += d;
    }

    ARIES_HOST_DEVICE_NO_INLINE Decimal &Decimal::operator+=(int16_t i) {
        Decimal d(i);
        return *this += d;
    }

    ARIES_HOST_DEVICE_NO_INLINE Decimal &Decimal::operator+=(int32_t i) {
        Decimal d(i);
        return *this += d;
    }

    ARIES_HOST_DEVICE_NO_INLINE Decimal &Decimal::operator+=(int64_t i) {
        Decimal d(i);
        return *this += d;
    }

    //unsigned
    ARIES_HOST_DEVICE_NO_INLINE Decimal &Decimal::operator+=(uint8_t i) {
        Decimal d(i);
        return *this += d;
    }

    ARIES_HOST_DEVICE_NO_INLINE Decimal &Decimal::operator+=(uint16_t i) {
        Decimal d(i);
        return *this += d;
    }

    ARIES_HOST_DEVICE_NO_INLINE Decimal &Decimal::operator+=(uint32_t i) {
        Decimal d(i);
        return *this += d;
    }

    ARIES_HOST_DEVICE_NO_INLINE Decimal &Decimal::operator+=(uint64_t i) {
        Decimal d(i);
        return *this += d;
    }

    //double / float
    ARIES_HOST_DEVICE_NO_INLINE double Decimal::operator+=(const float &f) {
        return *this += (double) f;
    }

    ARIES_HOST_DEVICE_NO_INLINE double Decimal::operator+=(const double &l) {
        return GetDouble() + l;
    }

    //self operator
    ARIES_HOST_DEVICE_NO_INLINE Decimal &Decimal::operator++() {
        Decimal d((int8_t) 1);
        *this += d;
        return *this;
    }

    ARIES_HOST_DEVICE_NO_INLINE Decimal Decimal::operator++(int32_t) {
        Decimal d((int8_t) 1);
        *this += d;
        return *this;
    }

    //signed
    ARIES_HOST_DEVICE_NO_INLINE Decimal operator+(const Decimal &left, const Decimal &right) {
        Decimal tmp(left);
        return tmp += right;
    }

    ARIES_HOST_DEVICE_NO_INLINE Decimal operator+(const Decimal &left, int8_t right) {
        Decimal tmp(left);
        return tmp += right;
    }

    ARIES_HOST_DEVICE_NO_INLINE Decimal operator+(const Decimal &left, int16_t right) {
        Decimal tmp(left);
        return tmp += right;
    }

    ARIES_HOST_DEVICE_NO_INLINE Decimal operator+(const Decimal &left, int32_t right) {
        Decimal tmp(left);
        return tmp += right;
    }

    ARIES_HOST_DEVICE_NO_INLINE Decimal operator+(const Decimal &left, int64_t right) {
        Decimal tmp(left);
        return tmp += right;
    }

    ARIES_HOST_DEVICE_NO_INLINE Decimal operator+(int8_t left, const Decimal &right) {
        Decimal tmp(left);
        return tmp += right;
    }

    ARIES_HOST_DEVICE_NO_INLINE Decimal operator+(int16_t left, const Decimal &right) {
        Decimal tmp(left);
        return tmp += right;
    }

    ARIES_HOST_DEVICE_NO_INLINE Decimal operator+(int32_t left, const Decimal &right) {
        Decimal tmp(left);
        return tmp += right;
    }

    ARIES_HOST_DEVICE_NO_INLINE Decimal operator+(int64_t left, const Decimal &right) {
        Decimal tmp(left);
        return tmp += right;
    }

    //unsigned
    ARIES_HOST_DEVICE_NO_INLINE Decimal operator+(const Decimal &left, uint8_t right) {
        Decimal tmp(left);
        return tmp += right;
    }

    ARIES_HOST_DEVICE_NO_INLINE Decimal operator+(const Decimal &left, uint16_t right) {
        Decimal tmp(left);
        return tmp += right;
    }

    ARIES_HOST_DEVICE_NO_INLINE Decimal operator+(const Decimal &left, uint32_t right) {
        Decimal tmp(left);
        return tmp += right;
    }

    ARIES_HOST_DEVICE_NO_INLINE Decimal operator+(const Decimal &left, uint64_t right) {
        Decimal tmp(left);
        return tmp += right;
    }

    ARIES_HOST_DEVICE_NO_INLINE Decimal operator+(uint8_t left, const Decimal &right) {
        Decimal tmp(left);
        return tmp += right;
    }

    ARIES_HOST_DEVICE_NO_INLINE Decimal operator+(uint16_t left, const Decimal &right) {
        Decimal tmp(left);
        return tmp += right;
    }

    ARIES_HOST_DEVICE_NO_INLINE Decimal operator+(uint32_t left, const Decimal &right) {
        Decimal tmp(left);
        return tmp += right;
    }

    ARIES_HOST_DEVICE_NO_INLINE Decimal operator+(uint64_t left, const Decimal &right) {
        Decimal tmp(left);
        return tmp += right;
    }

    //double / float
    ARIES_HOST_DEVICE_NO_INLINE double operator+(const Decimal &left, float right) {
        return left.GetDouble() + right;
    }

    ARIES_HOST_DEVICE_NO_INLINE double operator+(const Decimal &left, double right) {
        return left.GetDouble() + right;
    }

    ARIES_HOST_DEVICE_NO_INLINE double operator+(float left, const Decimal &right) {
        return left + right.GetDouble();
    }

    ARIES_HOST_DEVICE_NO_INLINE double operator+(double left, const Decimal &right) {
        return left + right.GetDouble();
    }

    // for sub
    ARIES_HOST_DEVICE_NO_INLINE void Decimal::CalcSubPrecision(const Decimal &d) {
        CalcAddPrecision(d);
    }

    ARIES_HOST_DEVICE_NO_INLINE void Decimal::CalcSubTargetPrecision(const Decimal &d) {
        CalcAddTargetPrecision(d);
    }

    ARIES_HOST_DEVICE_NO_INLINE void Decimal::CalcInnerSubPrecision( const Decimal &d ) {
        CalcInnerAddPrecision(d);
    }

    // op1 and op2 are positive
    ARIES_HOST_DEVICE_NO_INLINE int32_t Decimal::CompareInt(int32_t *op1, int32_t *op2) {
        int32_t res = 0;
        #pragma unroll
        for (int i = 0; i < NUM_TOTAL_DIG && res == 0; i++) {
            res = op1[i] - op2[i];
        }
        return res;
    }

    ARIES_HOST_DEVICE_NO_INLINE Decimal &Decimal::SubBothPositiveNums(Decimal &d) {
        int sign = 1;
        int32_t *p1 = (int32_t *) values, *p2 = (int32_t *) d.values;
        AlignAddSubData(d);
        int32_t r = CompareInt(p1, p2);
        if (r == 0) {
            #pragma unroll
            for (int i = 0; i < NUM_TOTAL_DIG; i++) {
                values[i] = 0;
            }
            return *this;
        } else if (r < 0) {
            int32_t *tmp;
            tmp = p1;
            p1 = p2;
            p2 = tmp;
            sign = -1;
        }
        //sub
        int32_t carry = 0; //借位
        for (int i = INDEX_LAST_DIG; i >= 0; i--) {
            p1[i] -= p2[i];
            p1[i] -= carry;
            if (p1[i] < 0) {
                p1[i] += PER_DEC_MAX_SCALE;
                carry = 1;
            } else {
                carry = 0;
            }
        }
        if (p1 != values) {
            #pragma unroll
            for (int i = 0; i < NUM_TOTAL_DIG; i++) {
                values[i] = p1[i];
            }
        }
        if (sign == -1) {
            Negate();
        }
        return *this;
    }

    ARIES_HOST_DEVICE_NO_INLINE Decimal &Decimal::operator-=(const Decimal &d) {
        CheckAndSetCalcPrecision();
        Decimal subd(d);
        subd.CheckAndSetCalcPrecision();
        //calculate precision after plus
        uint8_t intg0, frac0, mode0, error0;
        if (1 > 0) {
            Decimal calcPrecision(*this);
            calcPrecision.CalcAddPrecision(subd);
            calcPrecision.CalcInnerAddPrecision(subd);
            intg0 = calcPrecision.intg;
            frac0 = calcPrecision.frac;
            mode0 = calcPrecision.mode;
            error0 = calcPrecision.error;
        }
        bool subdNeg = subd.isLessZero();
        //
        if (isLessZero())   //被减数为负数
        {
            Negate();
            if (subdNeg) //减数为负数
            {
                // -a - -b = b - a = - (a - b)
                subd.Negate();
                SubBothPositiveNums(subd);
            } else //减数为正数
            {
                //-a - b = - (a + b)
                AddBothPositiveNums(subd);
            }
            Negate();
        } else   //被减数为正数
        {
            if (subdNeg) //减数为负数
            {
                //a - -b = a + b
                subd.Negate();
                AddBothPositiveNums(subd);
            } else {
                SubBothPositiveNums(subd);
            }
        }
        //set precision
        intg = intg0;
        frac = frac0;
        mode = mode0;
        error = error0;
        return *this;
    }

    //signed
    ARIES_HOST_DEVICE_NO_INLINE Decimal &Decimal::operator-=(int8_t i) {
        Decimal d(i);
        return *this -= d;
    }

    ARIES_HOST_DEVICE_NO_INLINE Decimal &Decimal::operator-=(int16_t i) {
        Decimal d(i);
        return *this -= d;
    }

    ARIES_HOST_DEVICE_NO_INLINE Decimal &Decimal::operator-=(int32_t i) {
        Decimal d(i);
        return *this -= d;
    }

    ARIES_HOST_DEVICE_NO_INLINE Decimal &Decimal::operator-=(int64_t i) {
        Decimal d(i);
        return *this -= d;
    }

    //unsigned
    ARIES_HOST_DEVICE_NO_INLINE Decimal &Decimal::operator-=(uint8_t i) {
        Decimal d(i);
        return *this -= d;
    }

    ARIES_HOST_DEVICE_NO_INLINE Decimal &Decimal::operator-=(uint16_t i) {
        Decimal d(i);
        return *this -= d;
    }

    ARIES_HOST_DEVICE_NO_INLINE Decimal &Decimal::operator-=(uint32_t i) {
        Decimal d(i);
        return *this -= d;
    }

    ARIES_HOST_DEVICE_NO_INLINE Decimal &Decimal::operator-=(uint64_t i) {
        Decimal d(i);
        return *this -= d;
    }

    //double / float
    ARIES_HOST_DEVICE_NO_INLINE double Decimal::operator-=(const float &f) {
        return GetDouble() - f;
    }

    ARIES_HOST_DEVICE_NO_INLINE double Decimal::operator-=(const double &l) {
        return GetDouble() - l;
    }

    //self operator
    ARIES_HOST_DEVICE_NO_INLINE Decimal &Decimal::operator--() {
        Decimal d((int8_t) 1);
        return *this -= d;
    }

    ARIES_HOST_DEVICE_NO_INLINE Decimal Decimal::operator--(int32_t) {
        Decimal tmp(*this);
        Decimal d((int8_t) 1);
        return tmp -= d;
    }

    ARIES_HOST_DEVICE_NO_INLINE Decimal operator-(const Decimal &left, const Decimal &right) {
        Decimal tmp(left);
        return tmp -= right;
    }

    ARIES_HOST_DEVICE_NO_INLINE Decimal operator-(const Decimal &left, int8_t right) {
        Decimal tmp(left);
        return tmp -= right;
    }

    ARIES_HOST_DEVICE_NO_INLINE Decimal operator-(const Decimal &left, int16_t right) {
        Decimal tmp(left);
        return tmp -= right;
    }

    ARIES_HOST_DEVICE_NO_INLINE Decimal operator-(const Decimal &left, int32_t right) {
        Decimal tmp(left);
        return tmp -= right;
    }

    ARIES_HOST_DEVICE_NO_INLINE Decimal operator-(const Decimal &left, int64_t right) {
        Decimal tmp(left);
        return tmp -= right;
    }

    ARIES_HOST_DEVICE_NO_INLINE Decimal operator-(int8_t left, const Decimal &right) {
        Decimal tmp(left);
        return tmp -= right;
    }

    ARIES_HOST_DEVICE_NO_INLINE Decimal operator-(int16_t left, const Decimal &right) {
        Decimal tmp(left);
        return tmp -= right;
    }

    ARIES_HOST_DEVICE_NO_INLINE Decimal operator-(int32_t left, const Decimal &right) {
        Decimal tmp(left);
        return tmp -= right;
    }

    ARIES_HOST_DEVICE_NO_INLINE Decimal operator-(int64_t left, const Decimal &right) {
        Decimal tmp(left);
        return tmp -= right;
    }

    //unsigned
    ARIES_HOST_DEVICE_NO_INLINE Decimal operator-(const Decimal &left, uint8_t right) {
        Decimal tmp(left);
        return tmp -= right;
    }

    ARIES_HOST_DEVICE_NO_INLINE Decimal operator-(const Decimal &left, uint16_t right) {
        Decimal tmp(left);
        return tmp -= right;
    }

    ARIES_HOST_DEVICE_NO_INLINE Decimal operator-(const Decimal &left, uint32_t right) {
        Decimal tmp(left);
        return tmp -= right;
    }

    ARIES_HOST_DEVICE_NO_INLINE Decimal operator-(const Decimal &left, uint64_t right) {
        Decimal tmp(left);
        return tmp -= right;
    }

    ARIES_HOST_DEVICE_NO_INLINE Decimal operator-(uint8_t left, const Decimal &right) {
        Decimal tmp(left);
        return tmp -= right;
    }

    ARIES_HOST_DEVICE_NO_INLINE Decimal operator-(uint16_t left, const Decimal &right) {
        Decimal tmp(left);
        return tmp -= right;
    }

    ARIES_HOST_DEVICE_NO_INLINE Decimal operator-(uint32_t left, const Decimal &right) {
        Decimal tmp(left);
        return tmp -= right;
    }

    ARIES_HOST_DEVICE_NO_INLINE Decimal operator-(uint64_t left, const Decimal &right) {
        Decimal tmp(left);
        return tmp -= right;
    }

    //double / float
    ARIES_HOST_DEVICE_NO_INLINE double operator-(const Decimal &left, const float right) {
        return left.GetDouble() - right;
    }

    ARIES_HOST_DEVICE_NO_INLINE double operator-(const Decimal &left, const double right) {
        return left.GetDouble() - right;
    }

    ARIES_HOST_DEVICE_NO_INLINE double operator-(const float left, const Decimal &right) {
        return left - right.GetDouble();
    }

    ARIES_HOST_DEVICE_NO_INLINE double operator-(const double left, const Decimal &right) {
        return left - right.GetDouble();
    }

    // for multiple
    ARIES_HOST_DEVICE_NO_INLINE void Decimal::CalcInnerMulPrecision(const Decimal& d) {
        uint16_t frac0 = aries_min(frac + d.frac, SUPPORTED_MAX_SCALE);
        uint16_t frac1, frac2;
        frac1 = frac2 = NEEDELEMENTS(frac0);
        uint16_t intg0 = intg + d.intg;
        uint16_t intg1, intg2;
        intg1 = intg2 = NEEDELEMENTS(intg0);
        uint8_t e = 0;
        FIX_INTG_FRAC_ERROR(INNER_MAX_PRECISION_INT32_NUM, intg1, frac1, e);
        SET_PREC_SCALE_VALUE(frac, frac0, frac1, frac2);
        SET_PREC_SCALE_VALUE(intg, intg0, intg1, intg2);
        SET_ERR(error, e);
    }

    ARIES_HOST_DEVICE_NO_INLINE void Decimal::CalcMulPrecision(const Decimal &d) {
        uint16_t frac0 = aries_min(GET_CALC_FRAC(error) + GET_CALC_FRAC(d.error), SUPPORTED_MAX_SCALE);
        uint16_t intg0 = GET_CALC_INTG(mode) + GET_CALC_INTG(d.mode);
        uint8_t e = 0;
        FIX_INTG_FRAC_ERROR(SUPPORTED_MAX_PRECISION, intg0, frac0, e);
        SET_CALC_INTG(mode, intg0);
        SET_CALC_FRAC(error,frac0);
        SET_ERR(error, e);
    }

    ARIES_HOST_DEVICE_NO_INLINE void Decimal::CalcMulTargetPrecision(const Decimal &d) {
        uint16_t frac0 = aries_min(frac + d.frac, SUPPORTED_MAX_SCALE);
        uint16_t intg0 = intg + d.intg;
        uint8_t e = 0;
        FIX_TAGET_INTG_FRAC_ERROR(SUPPORTED_MAX_PRECISION, intg0, frac0, e);
        intg = intg0;
        frac = frac0;
        error = e;
    }

    ARIES_HOST_DEVICE_NO_INLINE Decimal &Decimal::operator*=(const Decimal &d) {
        int sign = 1;
        CheckAndSetCalcPrecision();
        Decimal other(d);
        other.CheckAndSetCalcPrecision();
        if (isLessZero()) {
            sign = -sign;
            Negate();
        }
        if (other.isLessZero()) {
            sign = -sign;
            other.Negate();
        }
        int8_t cutFrac = NEEDELEMENTS(frac) + NEEDELEMENTS(d.frac);
        //calculate precision after multiple
        CalcMulPrecision(other);
        CalcInnerMulPrecision(other);
        cutFrac -= NEEDELEMENTS(frac);
        //swap values
        for ( int k = 0; k <= INDEX_LAST_DIG / 2; ++k ) {
            int32_t v = values[k];
            values[k] = values[INDEX_LAST_DIG - k];
            values[INDEX_LAST_DIG - k] = v;
            v = other.values[k];
            other.values[k] = other.values[INDEX_LAST_DIG - k];
            other.values[INDEX_LAST_DIG - k] = v;
        }
        int32_t res[NUM_TOTAL_DIG * 2] = {0};
        int32_t *op1 = values, *op2 = other.values;
        //multiple
        int32_t carry = 0;
        int64_t temp = 0;
        for (int i = 0; i < NUM_TOTAL_DIG; i++) {
            if (op2[i] == 0) {
                continue;
            }
            carry = 0;
            int32_t resIndex = 0;
            #pragma unroll
            for (int j = 0; j < NUM_TOTAL_DIG; j++) {
                resIndex = i + j;
                if (op1[j] || carry) {
                    if (op1[j]) {
                        temp = (int64_t) op1[j] * op2[i];
                    }
                    temp += res[resIndex] + carry;
                    if (temp >= PER_DEC_MAX_SCALE) {
                        carry = temp / PER_DEC_MAX_SCALE;
                        res[resIndex] = temp % PER_DEC_MAX_SCALE;
                    } else {
                        res[resIndex] = temp;
                        carry = 0;
                    }
                    temp = 0;
                }
            }
        }
        #pragma unroll
        for (int i = 0; i < NUM_TOTAL_DIG; i++) {
            values[INDEX_LAST_DIG - i] = res[i + cutFrac];
        }
        if (sign == -1) {
            Negate();
        }
        return *this;
    }

    //signed
    ARIES_HOST_DEVICE_NO_INLINE Decimal &Decimal::operator*=(int8_t i) {
        Decimal tmp(i);
        return *this *= tmp;
    }

    ARIES_HOST_DEVICE_NO_INLINE Decimal &Decimal::operator*=(int16_t i) {
        Decimal tmp(i);
        return *this *= tmp;
    }

    ARIES_HOST_DEVICE_NO_INLINE Decimal &Decimal::operator*=(int32_t i) {
        Decimal tmp(i);
        return *this *= tmp;
    }

    ARIES_HOST_DEVICE_NO_INLINE Decimal &Decimal::operator*=(int64_t i) {
        Decimal tmp(i);
        return *this *= tmp;
    }

    //unsigned
    ARIES_HOST_DEVICE_NO_INLINE Decimal &Decimal::operator*=(uint8_t i) {
        Decimal tmp(i);
        return *this *= tmp;
    }

    ARIES_HOST_DEVICE_NO_INLINE Decimal &Decimal::operator*=(uint16_t i) {
        Decimal tmp(i);
        return *this *= tmp;
    }

    ARIES_HOST_DEVICE_NO_INLINE Decimal &Decimal::operator*=(uint32_t i) {
        Decimal tmp(i);
        return *this *= tmp;
    }

    ARIES_HOST_DEVICE_NO_INLINE Decimal &Decimal::operator*=(uint64_t i) {
        Decimal tmp(i);
        return *this *= tmp;
    }

    //double / float
    ARIES_HOST_DEVICE_NO_INLINE double Decimal::operator*=(const float &f) {
        return GetDouble() * f;
    }

    ARIES_HOST_DEVICE_NO_INLINE double Decimal::operator*=(const double &d) {
        return GetDouble() * d;
    }

    //two operators
    ARIES_HOST_DEVICE_NO_INLINE Decimal operator*(const Decimal &left, const Decimal &right) {
        Decimal tmp(left);
        return tmp *= right;
    }

    //signed
    ARIES_HOST_DEVICE_NO_INLINE Decimal operator*(const Decimal &left, int8_t right) {
        Decimal tmp(right);
        return tmp *= left;
    }

    ARIES_HOST_DEVICE_NO_INLINE Decimal operator*(const Decimal &left, int16_t right) {
        Decimal tmp(right);
        return tmp *= left;
    }

    ARIES_HOST_DEVICE_NO_INLINE Decimal operator*(const Decimal &left, int32_t right) {
        Decimal tmp(right);
        return tmp *= left;
    }

    ARIES_HOST_DEVICE_NO_INLINE Decimal operator*(const Decimal &left, int64_t right) {
        Decimal tmp(right);
        return tmp *= left;
    }

    ARIES_HOST_DEVICE_NO_INLINE Decimal operator*(int8_t left, const Decimal &right) {
        Decimal tmp(left);
        return tmp *= right;
    }

    ARIES_HOST_DEVICE_NO_INLINE Decimal operator*(int16_t left, const Decimal &right) {
        Decimal tmp(left);
        return tmp *= right;
    }

    ARIES_HOST_DEVICE_NO_INLINE Decimal operator*(int32_t left, const Decimal &right) {
        Decimal tmp(left);
        return tmp *= right;
    }

    ARIES_HOST_DEVICE_NO_INLINE Decimal operator*(int64_t left, const Decimal &right) {
        Decimal tmp(left);
        return tmp *= right;
    }

    //unsigned
    ARIES_HOST_DEVICE_NO_INLINE Decimal operator*(const Decimal &left, uint8_t right) {
        Decimal tmp(right);
        return tmp *= left;
    }

    ARIES_HOST_DEVICE_NO_INLINE Decimal operator*(const Decimal &left, uint16_t right) {
        Decimal tmp(right);
        return tmp *= left;
    }

    ARIES_HOST_DEVICE_NO_INLINE Decimal operator*(const Decimal &left, uint32_t right) {
        Decimal tmp(right);
        return tmp *= left;
    }

    ARIES_HOST_DEVICE_NO_INLINE Decimal operator*(const Decimal &left, uint64_t right) {
        Decimal tmp(right);
        return tmp *= left;
    }

    ARIES_HOST_DEVICE_NO_INLINE Decimal operator*(uint8_t left, const Decimal &right) {
        Decimal tmp(left);
        return tmp *= right;
    }

    ARIES_HOST_DEVICE_NO_INLINE Decimal operator*(uint16_t left, const Decimal &right) {
        Decimal tmp(left);
        return tmp *= right;
    }

    ARIES_HOST_DEVICE_NO_INLINE Decimal operator*(uint32_t left, const Decimal &right) {
        Decimal tmp(left);
        return tmp *= right;
    }

    ARIES_HOST_DEVICE_NO_INLINE Decimal operator*(uint64_t left, const Decimal &right) {
        Decimal tmp(left);
        return tmp *= right;
    }

    //double / float
    ARIES_HOST_DEVICE_NO_INLINE double operator*(const Decimal &left, const float right) {
        return left.GetDouble() * right;
    }

    ARIES_HOST_DEVICE_NO_INLINE double operator*(const Decimal &left, const double right) {
        return left.GetDouble() * right;
    }

    ARIES_HOST_DEVICE_NO_INLINE double operator*(const float left, const Decimal &right) {
        return left * right.GetDouble();
    }

    ARIES_HOST_DEVICE_NO_INLINE double operator*(const double left, const Decimal &right) {
        return left * right.GetDouble();
    }

    // for division
    ARIES_HOST_DEVICE_NO_INLINE void Decimal::CalcInnerDivPrecision(const Decimal& d) {
        uint16_t frac0 = aries_min(frac + DIV_FIX_INNER_FRAC, SUPPORTED_MAX_SCALE);
        int highestV1, highestV2, prec1 = GetRealPrecision(highestV1), prec2 = d.GetRealPrecision(highestV2);
        int16_t intg0 = prec1 - frac - (prec2 - d.frac) + (highestV1 >= highestV2);
        if (intg0 < 0) {
            intg0 = 0;
        }
        uint16_t frac1, frac2;
        frac1 = frac2 = NEEDELEMENTS(frac0);
        uint16_t intg1, intg2;
        intg1 = intg2 = NEEDELEMENTS(intg0);
        uint8_t e = 0;
        FIX_INTG_FRAC_ERROR(INNER_MAX_PRECISION_INT32_NUM, intg1, frac1, e);
        SET_PREC_SCALE_VALUE(frac, frac0, frac1, frac2);
        SET_PREC_SCALE_VALUE(intg, intg0, intg1, intg2);
        SET_ERR(error, e);
    }

    ARIES_HOST_DEVICE_NO_INLINE void Decimal::CalcDivPrecision( const Decimal &d ) {
        uint16_t frac0 = aries_min(GET_CALC_FRAC(error) + DIV_FIX_EX_FRAC, SUPPORTED_MAX_SCALE);
        int highestV1, highestV2, prec1 = GetRealPrecision(highestV1), prec2 = d.GetRealPrecision(highestV2);
        int16_t intg0 = prec1 - GET_CALC_FRAC(error) - (prec2 - GET_CALC_FRAC(d.error)) + (highestV1 >= highestV2);
        if (intg0 < 0) {
            intg0 = 0;
        }
        uint8_t e = 0;
        FIX_INTG_FRAC_ERROR(SUPPORTED_MAX_PRECISION, intg0, frac0, e);
        SET_CALC_INTG(mode, intg0);
        SET_CALC_FRAC(error,frac0);
        SET_ERR(error, e);
    }

    ARIES_HOST_DEVICE_NO_INLINE void Decimal::CalcDivTargetPrecision( const Decimal &d ) {
        uint16_t frac0 = aries_min(frac + DIV_FIX_EX_FRAC, SUPPORTED_MAX_SCALE);
        uint16_t intg0 = aries_min(intg + d.frac, SUPPORTED_MAX_PRECISION);
        uint8_t e = 0;
        FIX_TAGET_INTG_FRAC_ERROR(SUPPORTED_MAX_PRECISION, intg0, frac0, e);
        intg = intg0;
        frac = frac0;
        error = e;
    }

    ARIES_HOST_DEVICE_NO_INLINE Decimal &Decimal::operator>>(int n) {
        int shiftDigits = n % DIG_PER_INT32;
        int shiftInt = n / DIG_PER_INT32;
        if (shiftDigits) {
            int lower = GetPowers10(shiftDigits);
            int higher = GetPowers10( DIG_PER_INT32 - shiftDigits);
            int carry = 0, temp = 0;
            #pragma unroll
            for (int i = 0; i < NUM_TOTAL_DIG; i++) {
                if (values[i] != 0) {
                    temp = values[i] % lower;
                    values[i] = values[i] / lower;
                } else {
                    temp = 0;
                }
                if (carry) {
                    values[i] += carry * higher;
                }
                carry = temp;
            }
        }
        if (shiftInt) {
            for (int i = INDEX_LAST_DIG; i >= shiftInt; i--) {
                values[i] = values[i - shiftInt];
            }
            for (int i = 0; i < shiftInt; i++) {
                values[i] = 0;
            }
        }
        //for check
        for (int i = 0; i < shiftInt; i++) {
            assert(values[i] == 0);
        }
        if (shiftDigits) {
            int lower = GetPowers10(shiftDigits);
            assert(values[shiftInt] / lower == 0);
        }
        return *this;
    }

    ARIES_HOST_DEVICE_NO_INLINE Decimal &Decimal::operator<<(int n) {
        int shiftDigits = n % DIG_PER_INT32;
        int shiftInt = n / DIG_PER_INT32;
        int lower = GetPowers10( DIG_PER_INT32 - shiftDigits);
        int higher = GetPowers10(shiftDigits);
        if (shiftDigits) {
            int carry = 0, temp = 0;
            for (int i = INDEX_LAST_DIG; i >= 0; i--) {
                if (values[i] != 0) {
                    temp = values[i] / lower;
                    values[i] = values[i] % lower * higher;
                } else {
                    temp = 0;
                }
                if (carry) {
                    values[i] += carry;
                }
                carry = temp;
            }
        }
        if (shiftInt) {
            for (int i = 0; i < NUM_TOTAL_DIG - shiftInt; i++) {
                values[i] = values[i + shiftInt];
            }
            for (int i = NUM_TOTAL_DIG - shiftInt; i < NUM_TOTAL_DIG; i++) {
                values[i] = 0;
            }
        }
        intg += n;
        return *this;
    }

    ARIES_HOST_DEVICE_NO_INLINE void Decimal::UpdateIntgDigits() {
        int validPos = 0;
        for ( validPos = 0; validPos < NUM_TOTAL_DIG; ++validPos )
        {
            if (values[validPos]) {
                break;
            }
        }
        int intg0 = NUM_TOTAL_DIG - validPos - NEEDELEMENTS(frac);
        if (intg0 <= 0) {
            intg = 0;
        } else {
            int v = aries_abs(values[validPos]);
            int digit = 1;
            while(v >= GetPowers10(digit) && ++digit < DIG_PER_INT32);
            intg = (intg0 - 1) * DIG_PER_INT32 + digit;
        }
    }

    ARIES_HOST_DEVICE_NO_INLINE int Decimal::GetRealPrecision(int &highestValue) const {
        int validPos = 0;
        for ( ; validPos < NUM_TOTAL_DIG; ++validPos )
        {
            if (values[validPos]) {
                break;
            }
        }
        int prec0 = NUM_TOTAL_DIG - validPos;
        if (prec0 <= 0) {
            highestValue = 0;
            return 0;
        }
        int v = aries_abs(values[validPos]);
        int digit = 1;
        while(v >= GetPowers10(digit) && ++digit < DIG_PER_INT32);
        highestValue = v / GetPowers10(digit - 1);
        if (frac == 0) {
            return digit + (prec0 - 1) * DIG_PER_INT32;
        } else {
            int lastFrac = frac % DIG_PER_INT32;
            return digit + (prec0 - 2) * DIG_PER_INT32 + (lastFrac == 0 ? DIG_PER_INT32 :  lastFrac);
        }
    }

    ARIES_HOST_DEVICE_NO_INLINE void Decimal::CheckAndSetCalcPrecision() {
        CheckAndSetRealPrecision();
        if (GET_CALC_FRAC(error) == 0 && GET_CALC_INTG(mode) == 0) {
            SET_CALC_FRAC(error, frac);
            SET_CALC_INTG(mode, intg);
        }
    }

    ARIES_HOST_DEVICE_NO_INLINE void Decimal::CheckAndSetRealPrecision() {
        int highest;
        int prec = GetRealPrecision(highest);
        intg = prec - frac;
        if ((intg & 0x80) > 0) {
            intg = 0;
        }
        if (intg == 0) {
            intg = 1;
        }
    }

    ARIES_HOST_DEVICE_NO_INLINE int Decimal::GetRealIntgSize(int &highestValue) const {
        int validPos = 0;
        for ( ; validPos < NUM_TOTAL_DIG; ++validPos )
        {
            if (values[validPos]) {
                break;
            }
        }
        int intg0 = NUM_TOTAL_DIG - validPos - NEEDELEMENTS(frac);
        if (intg0 <= 0) {
            highestValue = 0;
            intg0 = 0;
        } else {
            highestValue = values[validPos];
        }
        return intg0;
    }

    ARIES_HOST_DEVICE_NO_INLINE Decimal &Decimal::GenIntDecimal(int shift) {
        int n = shift;
        if (frac) {
            n -= DIG_PER_INT32 - frac % DIG_PER_INT32;
        }
        if (n > 0) {
            *this << n;
        } else if (n < 0) {
            *this >> (-n);
        }
        frac = 0;
        UpdateIntgDigits();
        return *this;
    }

    ARIES_HOST_DEVICE_NO_INLINE Decimal Decimal::HalfIntDecimal(const Decimal d1, const Decimal d2) {
        Decimal tmp(d1);
        tmp += d2;
        int32_t rds = 0;
        int64_t t[NUM_TOTAL_DIG];
        #pragma unroll
        for (int i = 0; i < NUM_TOTAL_DIG; i++) {
            t[i] = tmp.values[i];
        }
        #pragma unroll
        for (int i = 0; i < NUM_TOTAL_DIG; i++) {
            if (rds) {
                t[i] += rds * PER_DEC_MAX_SCALE;
            }
            if (t[i]) {
                rds = t[i] % 2;
                t[i] /= 2;
            }
        }
        #pragma unroll
        for (int i = 0; i < NUM_TOTAL_DIG; i++) {
            tmp.values[i] = t[i];
        }
        return tmp;
    }

    ARIES_HOST_DEVICE_NO_INLINE Decimal Decimal::DivInt(const Decimal ds, const Decimal dt, Decimal &residuel) {
        if (ds.isZero()) {
            residuel = 0;
            return ds;
        }
        int q = ds.intg - dt.intg;
        Decimal qmax(q + 1, 0), qmin(q, 0), qmid, rsdmax, rsdmin, rsdtmp;
        qmax.GenMaxDecByPrecision();
        qmin.GenMinDecByPrecision();
        Decimal t = qmax * dt;
        rsdmax = ds - t;
        if (rsdmax >= 0) {
            residuel = rsdmax;
            return qmax;
        }
        rsdmin = ds - qmin * dt;
        if (rsdmin == 0) {
            residuel = 0;
            return qmin;
        }
        assert(rsdmin > 0);
        while (qmin < qmax) {
            qmid = HalfIntDecimal(qmax, qmin);
            if (qmid == qmin) {
                break;
            }
            rsdtmp = ds - qmid * dt;
            if (rsdtmp == 0) {
                rsdmin = 0;
                qmin = qmid;
                break;
            } else if (rsdtmp > 0) {
                rsdmin = rsdtmp;
                qmin = qmid;
            } else {
                rsdmax = rsdtmp;
                qmax = qmid;
            }
        }
        residuel = rsdmin;
        return qmin;
    }

    ARIES_HOST_DEVICE_NO_INLINE Decimal& Decimal::DivByInt(const Decimal &d, int shift, bool isMod) {
        int dvt = d.values[INDEX_LAST_DIG];
        int remainder = 0;
        *this << shift;
        #pragma unroll
        for (int i = 0; i < NUM_TOTAL_DIG; ++i) {
            if (values[i] || remainder) {
                int64_t tmp = (int64_t) values[i] + (int64_t) remainder * PER_DEC_MAX_SCALE;
                values[i] = tmp / dvt;
                remainder = tmp % dvt;
            }
        }
        if (isMod) {
            *this = remainder;
        } else if (remainder << 1 > dvt) {
            *this += 1;
        }
        return *this;
    }

    ARIES_HOST_DEVICE_NO_INLINE Decimal& Decimal::DivByInt64(const Decimal &divisor, int shift, bool isMod) {
        int64_t dvs = ToInt64();
        while (shift > DIG_PER_INT32) {
            dvs *= GetPowers10(DIG_PER_INT32);
            shift -= DIG_PER_INT32;
        }
        dvs *= GetPowers10(shift);
        int64_t dvt = divisor.ToInt64();
        int64_t res = isMod ? (dvs % dvt) : (dvs / dvt + (((dvs % dvt) << 1) >= dvt ? 1 : 0));
        return *this = res;
    }

    ARIES_HOST_DEVICE_NO_INLINE Decimal &Decimal::Negate() {
        #pragma unroll
        for (int i = 0; i < NUM_TOTAL_DIG; i++) {
            values[i] = -values[i];
        }
        return *this;
    }

    ARIES_HOST_DEVICE_NO_INLINE Decimal &Decimal::IntToFrac(int fracDigits) {
        int frac0 = NEEDELEMENTS(fracDigits);

        Decimal intgPart(*this);
        intgPart >> (fracDigits);
        Decimal fracPart(*this);
        fracPart << ( DIG_PER_INT32 * NUM_TOTAL_DIG - fracDigits);
        for (int i = 0; i < NUM_TOTAL_DIG - frac0; i++) {
            values[i] = intgPart.values[i + frac0];
        }
        int fracBase = NUM_TOTAL_DIG - frac0;
        for (int i = fracBase; i < NUM_TOTAL_DIG; i++) {
            values[i] = fracPart.values[i - fracBase];
        }
        frac = fracDigits;
        UpdateIntgDigits();
        return *this;
    }

    ARIES_HOST_DEVICE_NO_INLINE void Decimal::CopyValue(Decimal &d) {
        #pragma unroll
        for (int i = 0; i < NUM_TOTAL_DIG; ++i) {
            values[i] = d.values[i];
        }
    }

    ARIES_HOST_DEVICE_NO_INLINE Decimal& Decimal::DivOrMod( const Decimal &d, bool isMod ) {
#ifdef COMPUTE_BY_STRING
        char divitend[128] =
        {   0};
        char divisor[128] =
        {   0};
        char result[128] =
        {   0};
        GetDivDecimalStr( divitend );
        Decimal tmpDt( d );
        tmpDt.GetDivDecimalStr( divisor );
        //multiple
        int len = aries_strlen( divitend );
        int end = len + d.frac + DIV_FIX_INNER_FRAC;
        for( int i = len; i < end; i++ )
        {
            divitend[i] = '0';
        }
        divitend[end] = 0;
        CalcDivPrecision( d );
        DivInt( divitend, divisor, 1, result );
        len = aries_strlen( result );
        assert( frac + intg >= len );
        if (len < frac)
        {
            for (int i = 0; i < frac -len; i++)
            {
                result[len + i] = '0';
            }
            result[frac] = 0;
            len = aries_strlen(result);
        }
        int p = len;
        InsertCh( result, len - frac, '.' );
        if (result[0] == '-')
        {
            p--;
        }
//        Decimal tmp( intg + frac, frac, result );
        Decimal tmp( p, frac, result );
        int err = error;
        *this = tmp;
        error = err;
#else
        CheckAndSetCalcPrecision();
        Decimal divitend(*this);
        Decimal divisor(d);
        divisor.CheckAndSetCalcPrecision();
        if (isMod)
        {
            CalcModPrecision(divisor);
            CalcInnerModPrecision(divisor);
        } else {
            CalcDivPrecision(divisor);
            CalcInnerDivPrecision(divisor);
        }
        if (isZero()) {
            return *this;
        } else if (d.isZero()) {
            SET_ERR(error, ERR_DIV_BY_ZERO);
            return *this;
        }

        uint8_t divitendFrac = divitend.frac;
        divitend.GenIntDecimal(isMod ? (divitendFrac < d.frac ? d.frac - divitendFrac : 0) : 0);
        int sign = 1;
        if (divitend.isLessZero()) {
            divitend.Negate();
            sign = -sign;
        }

        divisor.GenIntDecimal(isMod ? (d.frac < divitendFrac ? divitendFrac - d.frac : 0) : 0);
        if (divisor.isLessZero()) {
            sign = -sign;
            divisor.Negate();
        }
        int shift = d.frac + DIV_FIX_INNER_FRAC;
        if (!isMod) {
            // result is 0
            if (divitend.intg + shift < divisor.intg) {
                aries_memset(values, 0x00, sizeof(values));
                return *this;
            }
        } else {
            shift = 0;
        }

        Decimal res;
        //check if use integer div operator directly
        if (divitend.intg + shift <= DIG_PER_INT64 && divisor.intg <= DIG_PER_INT64) {
            res = divitend.DivByInt64(divisor, shift, isMod);
        } else if (divisor.intg <= DIG_PER_INT32) {
            res = divitend.DivByInt(divisor, shift, isMod);
        } else {
            int tmpEx = shift;
            int nDigits = 0;
            //one step DIG_PER_INT32 digit left
            Decimal tmpRes;
            if(shift == 0 && isMod){
                divitend.UpdateIntgDigits();
                nDigits = INNER_MAX_PRECISION - divitend.intg - 1;
                if (nDigits > tmpEx) {
                    nDigits = tmpEx;
                }
                tmpEx -= nDigits;
                divitend << (nDigits);
                tmpRes = DivInt(divitend, divisor, divitend);
                if (res != 0) {
                    res *= GetPowers10(nDigits);
                }
                res += tmpRes;
            }
            else{
                for (; tmpEx > 0;) {
                    divitend.UpdateIntgDigits();
                    nDigits = INNER_MAX_PRECISION - divitend.intg - 1;
                    if (nDigits > tmpEx) {
                        nDigits = tmpEx;
                    }
                    tmpEx -= nDigits;
                    divitend << (nDigits);
                    tmpRes = DivInt(divitend, divisor, divitend);
                    if (res != 0) {
                        res *= GetPowers10(nDigits);
                    }
                    res += tmpRes;
                }
            }
            //check if need round up
            if (isMod) {
                res = divitend;
            } else {
                if (divitend + divitend >= divisor) {
                    res += 1;
                }
            }
        }
        CopyValue(res.IntToFrac(frac));
        if (sign == -1) {
            Negate();
        }
#endif
        return *this;
    }

    ARIES_HOST_DEVICE_NO_INLINE Decimal &Decimal::operator/=(const Decimal &d) {
        return DivOrMod(d);
    }

    //signed
    ARIES_HOST_DEVICE_NO_INLINE Decimal &Decimal::operator/=(int8_t i) {
        Decimal d(i);
        return *this /= d;
    }

    ARIES_HOST_DEVICE_NO_INLINE Decimal &Decimal::operator/=(int16_t i) {
        Decimal d(i);
        return *this /= d;
    }

    ARIES_HOST_DEVICE_NO_INLINE Decimal &Decimal::operator/=(int32_t i) {
        Decimal d(i);
        return *this /= d;
    }

    ARIES_HOST_DEVICE_NO_INLINE Decimal &Decimal::operator/=(int64_t i) {
        Decimal d(i);
        return *this /= d;
    }

    //unsigned
    ARIES_HOST_DEVICE_NO_INLINE Decimal &Decimal::operator/=(uint8_t i) {
        Decimal d(i);
        return *this /= d;
    }

    ARIES_HOST_DEVICE_NO_INLINE Decimal &Decimal::operator/=(uint16_t i) {
        Decimal d(i);
        return *this /= d;
    }

    ARIES_HOST_DEVICE_NO_INLINE Decimal &Decimal::operator/=(uint32_t i) {
        Decimal d(i);
        return *this /= d;
    }

    ARIES_HOST_DEVICE_NO_INLINE Decimal &Decimal::operator/=(uint64_t i) {
        Decimal d(i);
        return *this /= d;
    }

    //double / float
    ARIES_HOST_DEVICE_NO_INLINE double Decimal::operator/=(const float &f) {
        return GetDouble() / f;
    }

    ARIES_HOST_DEVICE_NO_INLINE double Decimal::operator/=(const double &d) {
        return GetDouble() / d;
    }

    //two operators
    ARIES_HOST_DEVICE_NO_INLINE Decimal operator/(const Decimal &left, const Decimal &right) {
        Decimal tmp(left);
        return tmp /= right;
    }

    //signed
    ARIES_HOST_DEVICE_NO_INLINE Decimal operator/(const Decimal &left, int8_t right) {
        Decimal tmp(left);
        return tmp /= right;
    }

    ARIES_HOST_DEVICE_NO_INLINE Decimal operator/(const Decimal &left, int16_t right) {
        Decimal tmp(left);
        return tmp /= right;
    }

    ARIES_HOST_DEVICE_NO_INLINE Decimal operator/(const Decimal &left, int32_t right) {
        Decimal tmp(left);
        return tmp /= right;
    }

    ARIES_HOST_DEVICE_NO_INLINE Decimal operator/(const Decimal &left, int64_t right) {
        Decimal tmp(left);
        return tmp /= right;
    }

    ARIES_HOST_DEVICE_NO_INLINE Decimal operator/(int8_t left, const Decimal &right) {
        Decimal tmp(left);
        return tmp /= right;
    }

    ARIES_HOST_DEVICE_NO_INLINE Decimal operator/(int16_t left, const Decimal &right) {
        Decimal tmp(left);
        return tmp /= right;
    }

    ARIES_HOST_DEVICE_NO_INLINE Decimal operator/(int32_t left, const Decimal &right) {
        Decimal tmp(left);
        return tmp /= right;
    }

    ARIES_HOST_DEVICE_NO_INLINE Decimal operator/(int64_t left, const Decimal &right) {
        Decimal tmp(left);
        return tmp /= right;
    }

    //unsigned
    ARIES_HOST_DEVICE_NO_INLINE Decimal operator/(const Decimal &left, uint8_t right) {
        Decimal tmp(left);
        return tmp /= right;
    }

    ARIES_HOST_DEVICE_NO_INLINE Decimal operator/(const Decimal &left, uint16_t right) {
        Decimal tmp(left);
        return tmp /= right;
    }

    ARIES_HOST_DEVICE_NO_INLINE Decimal operator/(const Decimal &left, uint32_t right) {
        Decimal tmp(left);
        return tmp /= right;
    }

    ARIES_HOST_DEVICE_NO_INLINE Decimal operator/(const Decimal &left, uint64_t right) {
        Decimal tmp(left);
        return tmp /= right;
    }

    ARIES_HOST_DEVICE_NO_INLINE Decimal operator/(uint8_t left, const Decimal &right) {
        Decimal tmp(left);
        return tmp /= right;
    }

    ARIES_HOST_DEVICE_NO_INLINE Decimal operator/(uint16_t left, const Decimal &right) {
        Decimal tmp(left);
        return tmp /= right;
    }

    ARIES_HOST_DEVICE_NO_INLINE Decimal operator/(uint32_t left, const Decimal &right) {
        Decimal tmp(left);
        return tmp /= right;
    }

    ARIES_HOST_DEVICE_NO_INLINE Decimal operator/(uint64_t left, const Decimal &right) {
        Decimal tmp(left);
        return tmp /= right;
    }

    //double / float
    ARIES_HOST_DEVICE_NO_INLINE double operator/(const Decimal &left, const float right) {
        return left.GetDouble() / right;
    }

    ARIES_HOST_DEVICE_NO_INLINE double operator/(const Decimal &left, const double right) {
        return left.GetDouble() / right;
    }

    ARIES_HOST_DEVICE_NO_INLINE double operator/(const float left, const Decimal &right) {
        return left / right.GetDouble();
    }

    ARIES_HOST_DEVICE_NO_INLINE double operator/(const double left, const Decimal &right) {
        return left / right.GetDouble();
    }

    // for mod
    ARIES_HOST_DEVICE_NO_INLINE void Decimal::CalcModPrecision( const Decimal &d ) {
        int i = 0;
        uint8_t frac0 = GET_CALC_FRAC(error), frac1 = GET_CALC_FRAC(d.error), intg0;
        if (frac0 < frac1) {
            frac0 = frac1;
        } else {
            i = frac0 - frac1;
        }
        intg0 = GET_CALC_INTG(d.mode) + i;
        SET_CALC_INTG(mode, intg0);
        SET_CALC_FRAC(error, frac0);
    }

    ARIES_HOST_DEVICE_NO_INLINE void Decimal::CalcModTargetPrecision( const Decimal &d ) {
        int i = 0;
        uint8_t frac0 = frac, frac1 = d.frac, intg0;
        if (frac0 < frac1) {
            frac0 = frac1;
        } else {
            i = frac0 - frac1;
        }
        intg0 = d.intg + i;
        uint8_t e;
        FIX_TAGET_INTG_FRAC_ERROR(SUPPORTED_MAX_PRECISION, intg0, frac0, e);
        intg = intg0;
        frac = frac0;
        error = e;
    }

    ARIES_HOST_DEVICE_NO_INLINE void Decimal::CalcInnerModPrecision( const Decimal &d ) {
        int i = 0;
        if (frac < d.frac) {
            frac = d.frac;
        } else {
            i = frac - d.frac;
        }
        intg = d.intg + i;
    }

    ARIES_HOST_DEVICE_NO_INLINE Decimal &Decimal::operator%=(const Decimal& d) {
        return DivOrMod(d, true);
    }
    //signed
    ARIES_HOST_DEVICE_NO_INLINE Decimal &Decimal::operator%=(int8_t i) {
        Decimal d(i);
        return *this %= d;
    }

    ARIES_HOST_DEVICE_NO_INLINE Decimal &Decimal::operator%=(int16_t i) {
        Decimal d(i);
        return *this %= d;
    }

    ARIES_HOST_DEVICE_NO_INLINE Decimal &Decimal::operator%=(int32_t i) {
        Decimal d(i);
        return *this %= d;
    }

    ARIES_HOST_DEVICE_NO_INLINE Decimal &Decimal::operator%=(int64_t i) {
        Decimal d(i);
        return *this %= d;
    }

    //unsigned
    ARIES_HOST_DEVICE_NO_INLINE Decimal &Decimal::operator%=(uint8_t i) {
        Decimal d(i);
        return *this %= d;
    }

    ARIES_HOST_DEVICE_NO_INLINE Decimal &Decimal::operator%=(uint16_t i) {
        Decimal d(i);
        return *this %= d;
    }

    ARIES_HOST_DEVICE_NO_INLINE Decimal &Decimal::operator%=(uint32_t i) {
        Decimal d(i);
        return *this %= d;
    }

    ARIES_HOST_DEVICE_NO_INLINE Decimal &Decimal::operator%=(uint64_t i) {
        Decimal d(i);
        return *this %= d;
    }

    //double % float
    ARIES_HOST_DEVICE_NO_INLINE double Decimal::operator%=(const float &f) {
        return fmod(GetDouble(), f);
    }

    ARIES_HOST_DEVICE_NO_INLINE double Decimal::operator%=(const double &d) {
        return fmod(GetDouble(), d);
    }

    //two operators
    ARIES_HOST_DEVICE_NO_INLINE Decimal operator%(const Decimal &left, const Decimal &right) {
        Decimal tmp(left);
        return tmp %= right;
    }

    //signed
    ARIES_HOST_DEVICE_NO_INLINE Decimal operator%(const Decimal &left, int8_t right) {
        Decimal tmp(left);
        return tmp %= right;
    }

    ARIES_HOST_DEVICE_NO_INLINE Decimal operator%(const Decimal &left, int16_t right) {
        Decimal tmp(left);
        return tmp %= right;
    }

    ARIES_HOST_DEVICE_NO_INLINE Decimal operator%(const Decimal &left, int32_t right) {
        Decimal tmp(left);
        return tmp %= right;
    }

    ARIES_HOST_DEVICE_NO_INLINE Decimal operator%(const Decimal &left, int64_t right) {
        Decimal tmp(left);
        return tmp %= right;
    }

    ARIES_HOST_DEVICE_NO_INLINE Decimal operator%(int8_t left, const Decimal &right) {
        Decimal tmp(left);
        return tmp %= right;
    }

    ARIES_HOST_DEVICE_NO_INLINE Decimal operator%(int16_t left, const Decimal &right) {
        Decimal tmp(left);
        return tmp %= right;
    }

    ARIES_HOST_DEVICE_NO_INLINE Decimal operator%(int32_t left, const Decimal &right) {
        Decimal tmp(left);
        return tmp %= right;
    }

    ARIES_HOST_DEVICE_NO_INLINE Decimal operator%(int64_t left, const Decimal &right) {
        Decimal tmp(left);
        return tmp %= right;
    }

    //unsigned
    ARIES_HOST_DEVICE_NO_INLINE Decimal operator%(const Decimal &left, uint8_t right) {
        Decimal tmp(left);
        return tmp %= right;
    }

    ARIES_HOST_DEVICE_NO_INLINE Decimal operator%(const Decimal &left, uint16_t right) {
        Decimal tmp(left);
        return tmp %= right;
    }

    ARIES_HOST_DEVICE_NO_INLINE Decimal operator%(const Decimal &left, uint32_t right) {
        Decimal tmp(left);
        return tmp %= right;
    }

    ARIES_HOST_DEVICE_NO_INLINE Decimal operator%(const Decimal &left, uint64_t right) {
        Decimal tmp(left);
        return tmp %= right;
    }

    ARIES_HOST_DEVICE_NO_INLINE Decimal operator%(uint8_t left, const Decimal &right) {
        Decimal tmp(left);
        return tmp %= right;
    }

    ARIES_HOST_DEVICE_NO_INLINE Decimal operator%(uint16_t left, const Decimal &right) {
        Decimal tmp(left);
        return tmp %= right;
    }

    ARIES_HOST_DEVICE_NO_INLINE Decimal operator%(uint32_t left, const Decimal &right) {
        Decimal tmp(left);
        return tmp %= right;
    }

    ARIES_HOST_DEVICE_NO_INLINE Decimal operator%(uint64_t left, const Decimal &right) {
        Decimal tmp(left);
        return tmp %= right;
    }

    //double % float
    ARIES_HOST_DEVICE_NO_INLINE double operator%(const Decimal &left, const float right) {
        return fmod(left.GetDouble(), right);
    }

    ARIES_HOST_DEVICE_NO_INLINE double operator%(const Decimal &left, const double right) {
        return fmod(left.GetDouble(), right);
    }

    ARIES_HOST_DEVICE_NO_INLINE double operator%(const float left, const Decimal &right) {
        return fmod((double)left, right.GetDouble());
    }

    ARIES_HOST_DEVICE_NO_INLINE double operator%(const double left, const Decimal &right) {
        return fmod((double)left, right.GetDouble());
    }

    ARIES_HOST_DEVICE_NO_INLINE bool Decimal::isFracZero() const {
        for (int i = INDEX_LAST_DIG - NEEDELEMENTS(frac); i <= INDEX_LAST_DIG; ++i) {
            if (values[i]) {
                return false;
            }
        }
        return true;
    }

    ARIES_HOST_DEVICE_NO_INLINE bool Decimal::isZero() const {
        for (int i = 0; i < NUM_TOTAL_DIG; ++i) {
            if (values[i] != 0) {
                return false;
            }
        }
        return true;
    }

    ARIES_HOST_DEVICE_NO_INLINE bool Decimal::isLessZero() const {
        for (int i = 0; i < NUM_TOTAL_DIG; ++i) {
            if (values[i] < 0) {
                return true;
            }
        }
        return false;
    }

    ARIES_HOST_DEVICE_NO_INLINE bool Decimal::isLessEqualZero() const {
        for (int i = 0; i < NUM_TOTAL_DIG; ++i) {
            if (values[i] > 0) {
                return false;
            }
        }
        return true;
    }

    ARIES_HOST_DEVICE_NO_INLINE bool Decimal::isGreaterZero() const {
        for (int i = 0; i < NUM_TOTAL_DIG; ++i) {
            if (values[i] > 0) {
                return true;
            }
        }
        return false;
    }

    ARIES_HOST_DEVICE_NO_INLINE bool Decimal::isGreaterEqualZero() const {
        for (int i = 0; i < NUM_TOTAL_DIG; ++i) {
            if (values[i] < 0) {
                return false;
            }
        }
        return true;
    }

    ARIES_HOST_DEVICE_NO_INLINE int32_t Decimal::GetPowers10(int i) const {
        int32_t res = 1;
        switch (i) {
            case 0:
                res = 1;
                break;
            case 1:
                res = 10;
                break;
            case 2:
                res = 100;
                break;
            case 3:
                res = 1000;
                break;
            case 4:
                res = 10000;
                break;
            case 5:
                res = 100000;
                break;
            case 6:
                res = 1000000;
                break;
            case 7:
                res = 10000000;
                break;
            case 8:
                res = 100000000;
                break;
            case 9:
                res = PER_DEC_MAX_SCALE;
                break;
            default:
                break;
        }
        return res;
    }

    ARIES_HOST_DEVICE_NO_INLINE int32_t Decimal::GetFracMaxTable(int i) const {
        int32_t res = 0;
        switch (i) {
            case 0:
                res = 900000000;
                break;
            case 1:
                res = 990000000;
                break;
            case 2:
                res = 999000000;
                break;
            case 3:
                res = 999900000;
                break;
            case 4:
                res = 999990000;
                break;
            case 5:
                res = 999999000;
                break;
            case 6:
                res = 999999900;
                break;
            case 7:
                res = 999999990;
                break;
            case 8:
                res = 999999999;
                break;
            default:
                break;
        }
        return res;
    }

    ARIES_HOST_DEVICE_NO_INLINE void Decimal::GenMaxDecByPrecision() {
        int index = NUM_TOTAL_DIG - NEEDELEMENTS(intg) - NEEDELEMENTS(frac);
        // clear no use values
        for (int i = 0; i < index; i++) {
            values[i] = 0;
        }
        int firstDigits = intg % DIG_PER_INT32;
        if (firstDigits) {
            values[index++] = GetPowers10(firstDigits) - 1;
        }
        int32_t overPerDec = PER_DEC_MAX_SCALE - 1;
        for (int i = index; i < NUM_TOTAL_DIG; i++) {
            values[i] = overPerDec;
        }
        //replace last frac if necessary
        if (frac) {
            int lastDigits = frac % DIG_PER_INT32;
            if (lastDigits) {
                values[INDEX_LAST_DIG] = GetFracMaxTable(lastDigits - 1);
            }
        }
    }

    ARIES_HOST_DEVICE_NO_INLINE void Decimal::GenMinDecByPrecision() {
        int index = NUM_TOTAL_DIG - NEEDELEMENTS(intg) - NEEDELEMENTS(frac);
        // clear no use values
        for (int i = 0; i < index; i++) {
            values[i] = 0;
        }
        if (intg) {
            int firstDigits = intg % DIG_PER_INT32;
            if (firstDigits) {
                values[index++] = GetPowers10(firstDigits - 1);
            } else {
                values[index++] = GetPowers10( DIG_PER_INT32 - 1);
            }
        } else if (frac) {
            values[index++] = GetPowers10( DIG_PER_INT32 - 1);
        }
        for (int i = index; i < NUM_TOTAL_DIG; i++) {
            values[i] = 0;
        }
    }

    ARIES_HOST_DEVICE_NO_INLINE void Decimal::TransferData(const Decimal *v) {
        if (intg >= v->intg && frac >= v->frac) {
            SET_MODE(mode, GET_MODE(v->mode));
            SET_ERR(error, GET_ERR(v->error));
            int shift = NEEDELEMENTS(frac) - NEEDELEMENTS(v->frac);
            for (int i = shift; i < NUM_TOTAL_DIG; i++) {
                values[i - shift] = v->values[i];
            }
        } else {
            assert(0);
            SET_MODE(mode, GET_MODE(v->mode));
            SET_ERR(error, ERR_OVER_FLOW);
        }
        assert(intg + frac <= SUPPORTED_MAX_PRECISION && frac <= SUPPORTED_MAX_SCALE);
    }

    ARIES_HOST_DEVICE_NO_INLINE void Decimal::AlignAddSubData(Decimal &d) {
        if (frac == d.frac) {
            //do nothing
            return;
        }
        int fracc = NEEDELEMENTS(frac);
        int fracd = NEEDELEMENTS(d.frac);
        //align integer and frac part
        if (fracc == fracd) {
            //do nothing
            return;
        }
        if (fracc > fracd) {
            //shift forward d only, and discard high number
            int shift = fracc - fracd;
            for (int i = 0; i < NUM_TOTAL_DIG - shift; i++) {
                d.values[i] = d.values[i + shift];
            }
            for (int i = NUM_TOTAL_DIG - shift; i < NUM_TOTAL_DIG; i++) {
                d.values[i] = 0;
            }
        } else {
            //shift forward current only, and discard high number
            int shift = fracd - fracc;
            for (int i = 0; i < NUM_TOTAL_DIG - shift; i++) {
                values[i] = values[i + shift];
            }
            for (int i = NUM_TOTAL_DIG - shift; i < NUM_TOTAL_DIG; i++) {
                values[i] = 0;
            }
        }
    }

    ARIES_HOST_DEVICE_NO_INLINE void Decimal::initialize(uint32_t ig, uint32_t fc, uint32_t m) {
        if (fc > SUPPORTED_MAX_SCALE) {
            fc = SUPPORTED_MAX_SCALE;
        }
        if (ig + fc > SUPPORTED_MAX_PRECISION) {
            ig = SUPPORTED_MAX_PRECISION - fc;
        }
        intg = ig;
        frac = fc;
        mode = m;
        error = ERR_OK;
//        SET_CALC_INTG(mode, intg);
//        SET_CALC_FRAC(error, frac);
        aries_memset(values, 0x00, sizeof(values));
    }

    ARIES_HOST_DEVICE_NO_INLINE double Decimal::GetDouble() const {
        double z = 0;
        int frac0 = NEEDELEMENTS(frac);
        for (int i = 0; i < NUM_TOTAL_DIG - frac0; i++) {
            if (values[i]) {
                z += values[i];
            }
            if (z) {
                z *= PER_DEC_MAX_SCALE;
            }
        }
        //handle scale part
        double s = 0;
        for (int i = NUM_TOTAL_DIG - frac0; i < NUM_TOTAL_DIG; i++) {
            if (values[i]) {
                s += values[i];
            }
            if (s) {
                s *= PER_DEC_MAX_SCALE;
            }
        }
        for (int i = 0; i < frac0; i++) {
            s /= PER_DEC_MAX_SCALE;
        }
        z += s;
        return z / PER_DEC_MAX_SCALE;
    }

    ARIES_HOST_DEVICE_NO_INLINE int64_t Decimal::ToInt64() const {
        //only 2 digits are valid and no frac part
        int64_t res = values[INDEX_LAST_DIG];
        if (values[INDEX_LAST_DIG - 1]) {
            res += (int64_t) values[INDEX_LAST_DIG - 1] * PER_DEC_MAX_SCALE;
        }
        return res;
    }
    ARIES_HOST_DEVICE_NO_INLINE bool Decimal::CheckIfValidStr2Dec(char * str)
    {
        if (*str == '-') ++str;
        for ( int i = 0; i < aries_strlen(str); ++i )
        {
            if (aries_is_digit(str[i]))
            {
                continue;
            }
            if (str[i] == '.')
            {
                continue;
            }
            return false;
        }
        return true;
    }

    ARIES_HOST_DEVICE_NO_INLINE bool Decimal::CheckIfValidStr2Dec(char * str, int len)
    {
        if (*str == '-') ++str;
        for ( int i = 0; i < aries_strlen(str, len); ++i )
        {
            if (aries_is_digit(str[i]))
            {
                continue;
            }
            if (str[i] == '.')
            {
                continue;
            }
            return false;
        }
        return true;
    }

    ARIES_HOST_DEVICE_NO_INLINE bool Decimal::StringToDecimal( char * str, int len )
    {
        if (!CheckIfValidStr2Dec(str, len))
        {
            return false;
        }
        char sign = 1;
        if (*str == '-') {
            ++str;
            sign = -1;
        }
        char *intgend = aries_strchr(str, '.');
        int strLen = aries_strlen(str, len);
        int intgLen = intgend ? intgend - str : strLen;
        int fracLen = intgend ? strLen - intgLen - 1 : 0;
        assert(fracLen <= SUPPORTED_MAX_SCALE);
        assert(intgLen + fracLen <= SUPPORTED_MAX_PRECISION);
        intg = intgLen;
        frac = fracLen;
        SET_CALC_INTG(mode, intg);
        SET_CALC_FRAC(error, frac);
        int intg0 = NEEDELEMENTS(intgLen);
        int frac0 = NEEDELEMENTS(fracLen);
        int pos = NUM_TOTAL_DIG - frac0 - intg0;
        char temp[16];
        //handle intg part
        int firstLen = intgLen % DIG_PER_INT32;
        if (firstLen) {
            aries_strncpy(temp, str, firstLen);
            temp[firstLen] = 0;
            values[pos++] = aries_atoi(temp);
            str += firstLen;
        }
        for (int i = pos; i < NUM_TOTAL_DIG - frac0; i++) {
            aries_strncpy( temp, str, DIG_PER_INT32);
            temp[DIG_PER_INT32] = 0;
            values[i] = aries_atoi(temp);
            str += DIG_PER_INT32;
        }
        //handle frac part
        if (intgend) {
            str = intgend + 1;
            for (int i = NUM_TOTAL_DIG - frac0; i < NUM_TOTAL_DIG - 1; i++) {
                aries_strncpy( temp, str, DIG_PER_INT32);
                temp[DIG_PER_INT32] = 0;
                values[i] = aries_atoi(temp);
                str += DIG_PER_INT32;
            }
            //handle last one
            aries_strcpy(temp, str);
            values[INDEX_LAST_DIG] = aries_atoi(temp);
            int frac1 = fracLen % DIG_PER_INT32;
            if (frac1) {
                values[INDEX_LAST_DIG] *= GetPowers10( DIG_PER_INT32 - frac1);
            }
        }
        if (sign == -1) {
            #pragma unroll
            for (int i = 0; i < NUM_TOTAL_DIG; ++i) {
                values[i] = -values[i];
            }
        }
        return true;
    }

    /* mysql> select 999999999999999999999999999999999999999999999999999999999999999999999999999999999999;
       +--------------------------------------------------------------------------------------+
       | 999999999999999999999999999999999999999999999999999999999999999999999999999999999999 |
       +--------------------------------------------------------------------------------------+
       |                    99999999999999999999999999999999999999999999999999999999999999999 |
       +--------------------------------------------------------------------------------------+
       1 row in set, 1 warning (0.00 sec)

       mysql> show warnings;
       +---------+------+------------------------------------------------------------------------------------------------------------------------+
       | Level   | Code | Message                                                                                                                |
       +---------+------+------------------------------------------------------------------------------------------------------------------------+
       | Warning | 1292 | Truncated incorrect DECIMAL value: '999999999999999999999999999999999999999999999999999999999999999999999999999999999' |
       +---------+------+------------------------------------------------------------------------------------------------------------------------+
    */
    ARIES_HOST_DEVICE_NO_INLINE bool Decimal::StringToDecimal( char * str )
    {
        if (!CheckIfValidStr2Dec(str))
        {
            return false;
        }
        char sign = 1;
        if (*str == '-') {
            ++str;
            sign = -1;
        }
        char *intgend = aries_strchr(str, '.');
        int strLen = aries_strlen(str);
        int intgLen = intgend ? intgend - str : strLen;
        int fracLen = intgend ? strLen - intgLen - 1 : 0;
        assert(fracLen <= SUPPORTED_MAX_SCALE);
        assert(intgLen + fracLen <= SUPPORTED_MAX_PRECISION);
        intg = intgLen;
        frac = fracLen;
        SET_CALC_INTG(mode, intg);
        SET_CALC_FRAC(error, frac);
        int intg0 = NEEDELEMENTS(intgLen);
        int frac0 = NEEDELEMENTS(fracLen);
        int pos = NUM_TOTAL_DIG - frac0 - intg0;
        char temp[16];
        //handle intg part
        int firstLen = intgLen % DIG_PER_INT32;
        if (firstLen) {
            aries_strncpy(temp, str, firstLen);
            temp[firstLen] = 0;
            values[pos++] = aries_atoi(temp);
            str += firstLen;
        }
        for (int i = pos; i < NUM_TOTAL_DIG - frac0; i++) {
            aries_strncpy( temp, str, DIG_PER_INT32);
            temp[DIG_PER_INT32] = 0;
            values[i] = aries_atoi(temp);
            str += DIG_PER_INT32;
        }
        //handle frac part
        if (intgend) {
            str = intgend + 1;
            for (int i = NUM_TOTAL_DIG - frac0; i < NUM_TOTAL_DIG - 1; i++) {
                aries_strncpy( temp, str, DIG_PER_INT32);
                temp[DIG_PER_INT32] = 0;
                values[i] = aries_atoi(temp);
                str += DIG_PER_INT32;
            }
            //handle last one
            aries_strcpy(temp, str);
            values[INDEX_LAST_DIG] = aries_atoi(temp);
            int frac1 = fracLen % DIG_PER_INT32;
            if (frac1) {
                values[INDEX_LAST_DIG] *= GetPowers10( DIG_PER_INT32 - frac1);
            }
        }
        if (sign == -1) {
            #pragma unroll
            for (int i = 0; i < NUM_TOTAL_DIG; ++i) {
                values[i] = -values[i];
            }
        }
        return true;
    }

    //below methods are for computing long 10 based integer by char string
#ifdef COMPUTE_BY_STRING
    ARIES_HOST_DEVICE_NO_INLINE char* Decimal::GetDivDecimalStr( char *to)
    {
        int start = -1;
        for( int i = 0; i < NUM_TOTAL_DIG; i++ )
        {
            if (values[i] == 0)
            continue;
            start = i;
            break;
        }
        if( start == -1 )
        {
            aries_strcpy( to, "0");
        }
        else
        {
            aries_sprintf( to, "%d", values[start++] );
            char temp[16];
            for( int i = start; i < NUM_TOTAL_DIG - 1; i++ )
            {
                aries_sprintf( temp, values[i] < 0 ? "%010d" : "%09d", values[i] );
                aries_strcat( to, values[i] < 0 ? temp + 1 : temp );
            }
            //handle last one
            int remainLen = frac % DIG_PER_INT32;
            int end = NUM_TOTAL_DIG - 1;
            aries_sprintf( temp, values[end] < 0 ? "%010d" : "%09d", values[end] );
            aries_strncat( to, values[end] < 0 ? temp + 1 : temp, remainLen );
        }
        return to;
    }

    ARIES_HOST_DEVICE_NO_INLINE int Decimal::Compare( char *cmp1, char *cmp2)
    {
        size_t len1 = aries_strlen(cmp1), len2 = aries_strlen(cmp2);
        if (len1 > len2)
        {
            return 1;
        }
        else if (len1 < len2)
        {
            return -1;
        }
        else
        {
            return aries_strcmp(cmp1, cmp2);
        }
    }

    ARIES_HOST_DEVICE_NO_INLINE int Decimal::FindFirstNotOf( char *s, char ch)
    {
        char *p = s;
        if (ch)
        {
            while (*p && *p == ch) ++p;
        }
        return p - s;
    }

    ARIES_HOST_DEVICE_NO_INLINE char* Decimal::Erase( char *s, int startPos, int n)
    {
        int l = aries_strlen(s);
        if (l <= startPos || n <= 0)
        {
            return s;
        }
        int endPos = startPos + n;
        if (l <= endPos)
        {
            s[startPos] = 0;
        }
        else
        {
            aries_strcpy(s + startPos, s + endPos);
        }
        return s;
    }

    ARIES_HOST_DEVICE_NO_INLINE char* Decimal::DivInt(char *str1, char *str2, int mode, char * result)
    {
        char quotient[128] =
        {   0}, residue[128] =
        {   0};   //定义商和余数
        int signds = 1, signdt = 1;
        if (*str2 == '0')//判断除数是否为0
        {
            error = ERR_DIV_BY_ZERO;
            aries_strcpy(result, "ERROR!");
            return result;
        }
        if (*str1 == '0')     //判断被除数是否为0
        {
            aries_strcpy(quotient, "0");
            aries_strcpy(residue, "0");
        }
        if (str1[0] == '-')
        {
            ++str1;
            signds *= -1;
            signdt = -1;
        }
        if (str2[0] == '-')
        {
            ++str2;
            signds *= -1;
        }
        int res = Compare(str1, str2);
        if (res < 0)
        {
            aries_strcpy(quotient, "0");
            aries_strcpy(residue, str1);
        }
        else if (res == 0)
        {
            aries_strcpy(quotient, "1");
            aries_strcpy(residue, "0");
        }
        else
        {
            int divitendLen = aries_strlen(str1), divisorLen = aries_strlen(str2);
            char tempstr[128] =
            {   0};
            aries_strncpy(tempstr, str1, divisorLen - 1);
            tempstr[divisorLen] = 0;
            int len = 0;
            //模拟手工除法竖式
            for (int i = divisorLen - 1; i < divitendLen; i++)
            {
                len = aries_strlen(tempstr);
                tempstr[len] = str1[i];
                tempstr[len + 1] = 0;
                Erase(tempstr, 0, FindFirstNotOf(tempstr, '0'));
                if (aries_strlen(tempstr) == 0)
                {
                    aries_strcpy(tempstr, "0");
                }
                for (char ch = '9'; ch >= '0'; ch--) //试商
                {
                    char temp[16];
                    temp[0] = ch;
                    temp[1] = 0;
                    char r[128] =
                    {   0};
                    if( Compare( MulInt( (char *)str2, (char *)temp, r), tempstr ) <= 0 )
                    {
                        len = aries_strlen(quotient);
                        quotient[len] = ch;
                        quotient[len + 1] = 0;
                        SubInt( tempstr, MulInt( str2, temp, r ) , tempstr);
                        break;
                    }
                }
            }
            aries_strcpy(residue, tempstr);
        }
        //去除结果中的前导0
        Erase(quotient, 0, FindFirstNotOf(quotient, '0'));
        if (aries_strlen(quotient) == 0)
        {
            aries_strcpy(quotient, "0");
        }
        if ((signds == -1) && (quotient[0] != '0'))
        {
            InsertCh(quotient, 0, '-');
        }
        if ((signdt == -1) && (residue[0] != '0'))
        {
            InsertCh(residue, 0, '-');
        }
        if (mode == 1)
        {
            aries_strcpy(result, quotient);
        }
        else
        {
            aries_strcpy(result, residue);
        }
        return result;
    }

    ARIES_HOST_DEVICE_NO_INLINE char* Decimal::MulInt(char *str1, char *str2, char * result)
    {
        int sign = 1;
        char str[128] =
        {   0};  //记录当前值
        str[0] = '0';
        if (str1[0] == '-')
        {
            sign *= -1;
            str1++;
        }
        if (str2[0] == '-')
        {
            sign *= -1;
            str2++;
        }
        int i, j;
        size_t L1 = aries_strlen(str1), L2 = aries_strlen(str2);
        for (i = L2 - 1; i >= 0; i--)              //模拟手工乘法竖式
        {
            char tempstr[128] =
            {   0};
            int int1 = 0, int2 = 0, int3 = int(str2[i]) - '0';
            if (int3 != 0)
            {
                for (j = 1; j <= (int)(L2 - 1 - i); j++)
                {
                    tempstr[j - 1] = 0;
                }
                for (j = L1 - 1; j >= 0; j--)
                {
                    int1 = (int3*(int(str1[j]) - '0') + int2) % 10;
                    int2 = (int3*(int(str1[j]) - '0') + int2) / 10;
                    InsertCh(tempstr, 0, char(int1 + '0'));
                }
                if (int2 != 0)
                {
                    InsertCh(tempstr, 0, char(int2 + '0'));
                }
            }
            AddInt(str, tempstr, str);
        }
        //去除结果中的前导0
        Erase(str, 0, FindFirstNotOf(str, '0'));
        if (aries_strlen(str) == 0)
        {
            aries_strcpy(str, "0");
        }
        if ((sign == -1) && (str[0] != '0'))
        {
            InsertCh(str, 0, '-');
        }

        aries_strcpy(result, str);
        return result;
    }

    ARIES_HOST_DEVICE_NO_INLINE char* Decimal::SubInt(char *str1, char *str2, char *result)
    {
        int sign = 1; //sign为符号位
        int i, j;
        if (str2[0] == '-')
        {
            result = AddInt(str1, str2 + 1, result);
        }
        else
        {
            int res = Compare(str1, str2);
            if (res == 0)
            {
                aries_strcpy(result, "0");
                return result;
            }
            if (res < 0)
            {
                sign = -1;
                char *temp = str1;
                str1 = str2;
                str2 = temp;
            }
            int len1 = aries_strlen(str1), len2 = aries_strlen(str2);
            int tmplen = len1 - len2;
            for (i = len2 - 1; i >= 0; i--)
            {
                if (str1[i + tmplen] < str2[i])          //借位
                {
                    j = 1;
                    while (1)
                    {
                        if (str1[tmplen - j + i] == '0')
                        {
                            str1[i + tmplen - j] = '9';
                            j++;
                        }
                        else
                        {
                            str1[i + tmplen - j] = char(int(str1[i + tmplen - j]) - 1);
                            break;
                        }
                    }
                    result[i + tmplen] = char(str1[i + tmplen] - str2[i] + ':');
                }
                else
                {
                    result[i + tmplen] = char(str1[i + tmplen] - str2[i] + '0');
                }
            }
            for (i = tmplen - 1; i >= 0; i--)
            result[i] = str1[i];
        }
        //去出结果中多余的前导0
        Erase(result, 0, FindFirstNotOf(result, '0'));
        if (aries_strlen(result) == 0)
        {
            aries_strcpy(result, "0");
        }
        if ((sign == -1) && (result[0] != '0'))
        {
            InsertCh(result, 0, '-');
        }
        return result;
    }

    ARIES_HOST_DEVICE_NO_INLINE char* Decimal::AddInt(char *str1, char *str2, char *result)
    {
        int sign = 1;          //sign为符号为
        char str[128] =
        {   0};
        if (str1[0] == '-')
        {
            if (str2[0] == '-')       //负负
            {
                sign = -1;
                AddInt(str1 + 1, str2 + 1, str);       //去掉正负号
            }
            else             //负正
            {
                SubInt(str2, str1 + 1, str);
            }
        }
        else
        {
            if (str2[0] == '-')        //正负
            {
                SubInt(str1, str2 + 1, str);
            }
            else                    //正正，把两个整数对齐，短整数前面加0补齐
            {
                int L1 = aries_strlen(str1), L2 = aries_strlen(str2);
                int i, l;
                char tmp[128];
                if (L1 < L2)
                {
                    l = L2 - L1;
                    for (i = 0; i < l; i++)
                    {
                        tmp[i] = '0';
                    }
                    tmp[l] = 0;
                    InsertStr(str1, 0, tmp);
                }
                else
                {
                    l = L1 - L2;
                    for (i = 0; i < L1 - L2; i++)
                    {
                        tmp[i] = '0';
                    }
                    tmp[l] = 0;
                    InsertStr(str2, 0, tmp);
                }
                int int1 = 0, int2 = 0; //int2记录进位
                l = aries_strlen(str1);
                for (i = l - 1; i >= 0; i--)
                {
                    int1 = (int(str1[i]) - '0' + int(str2[i]) - '0' + int2) % 10;
                    int2 = (int(str1[i]) - '0' + int(str2[i]) - '0' + int2) / 10;
                    str[i + 1] = char(int1 + '0');
                }
                str[l + 1] = 0;
                if (int2 != 0)
                {
                    result[0] = char(int2 + '0');
                }
                else
                {
                    aries_strcpy(str, str + 1);
                }
            }
        }
        //运算符处理符号
        if ((sign == -1) && (str[0] != '0'))
        {
            InsertCh(str, 0, '-');
        }
        aries_strcpy(result, str);
        return result;
    }

    ARIES_HOST_DEVICE_NO_INLINE char* Decimal::InsertStr(char *str, int pos, char *in)
    {
        int len = aries_strlen(str);
        int inLen = aries_strlen(in);
        assert(len + inLen < 128);
        int insertPos = len < pos ? len : pos;
        if (len == insertPos)
        {
            aries_strcat(str, in);
        }
        else
        {
            char tmp[128];
            aries_strcpy(tmp, str + insertPos);
            aries_strcpy(str + insertPos, in);
            aries_strcpy(str + insertPos + inLen, tmp);
        }
        return str;
    }

    ARIES_HOST_DEVICE_NO_INLINE char* Decimal::InsertCh(char *str, int pos, char in)
    {
        char temp[8];
        temp[0] = in;
        temp[1] = 0;
        return InsertStr(str, pos, temp);
    }
#endif

END_ARIES_ACC_NAMESPACE

